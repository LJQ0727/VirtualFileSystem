﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS,
              FCB *start_of_fcb)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
  fs->STORAGE_BLOCK_COUNT = (fs->STORAGE_SIZE - fs->SUPERBLOCK_SIZE - (fs->FCB_SIZE * fs->FCB_ENTRIES)) / fs->STORAGE_BLOCK_SIZE;

  // init some custom pointers
  fs->start_of_fcb = start_of_fcb;
  fs->start_of_superblock = volume;
  fs->start_of_contents = volume + FILE_BASE_ADDRESS;
  fs->cwd = -1;   // the root directory will have parent dir index -1

  // initialize volume
  for (int i = 0; i < VOLUME_SIZE; i++)
    volume[i] = 0;

  // initialize fcb
  for (int i = 0; i < FCB_ENTRIES; i++) {
    fs->start_of_fcb[i].is_dir = false;
    fs->start_of_fcb[i].is_on = false;
    fs->start_of_fcb[i].parent_dir_idx = -1;
  }

  // make root directory and cd to it
  fs_gsys(fs, MKDIR, "/\0");
  fs->cwd = 0;  // root will be created at idx 0
}

__device__ bool strmatch(char *start1, char* start2);

__device__ uchar * get_content(FileSystem *fs, int block_idx, int byte_offset) {
  // given a block index, get the pointer to the content of a file (or directory)
  return fs->start_of_contents + block_idx * fs->STORAGE_BLOCK_SIZE + byte_offset;
}

__device__ void set_gtime_recursive(FileSystem *fs, int fcb_idx, u32 gtime) {
  // recursively set the gtime of the fcb and propagate this change to all parent dirs
  FCB *fcb = fs->start_of_fcb + fcb_idx;
  fcb->modified_time = gtime;
  printf("modified gtime of '%s' to %d, isdir: %d, parent_dir_idx: %d\n", fcb->filename , gtime,fcb->is_dir, fcb->parent_dir_idx);
  if (fcb->dir_idx != -1 && !fcb->is_dir) {
    // set gtime for the file's dir
    set_gtime_recursive(fs, fcb->dir_idx, gtime);
  }
  if (fcb->parent_dir_idx != -1) {
    set_gtime_recursive(fs, fcb->parent_dir_idx, gtime);
  }
}
  

__device__ bool strmatch(char *start1, char* start2) {
  // match two strings, return true if they are the same
  int idx = 0;
  while (!(start1[idx] == '\0' || start2[idx] == '\0'))
  {
    if (start1[idx] != start2[idx])
    {
      return false;
    }
    idx++;
  }
  if (start1[idx] != start2[idx])
  {
    return false;
  }

  return true;
}

__device__ void mark_block_used(FileSystem *fs, int block_idx) {
  // mark a block as used in the superblock
  // operate on only one block at a time
  uchar bitmap = fs->start_of_superblock[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  fs->start_of_superblock[block_idx/8] = bitmap | mask;
}

__device__ void mark_block_unused(FileSystem *fs, int block_idx) {
  // mark a block as unused in the superblock
  // operate on only one block at a time
  uchar bitmap = fs->start_of_superblock[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  fs->start_of_superblock[block_idx/8] = bitmap & ~mask;
}

__device__ bool check_block_used(FileSystem *fs, int block_idx) {
  uchar bitmap = fs->start_of_superblock[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  return bitmap & mask;
}


__device__ void my_memcpy(char *destination, char *source, int size) {
  // this will not automatically add '\0' to the string
  for (int i = 0; i < size; i++)
  {
    destination[i] = source[i];
  }
  
}

__device__ int my_strlen(char *s) {
  // find the length of a string, **including '\0'**
  int idx = 0;
  while (s[idx] != '\0')
  {
    idx++;
  }
  return idx+1;
}

__device__ FCB * get_file_in_curr_dir(FileSystem *fs, char* s) {
  // return the pointer to the file or subdir if the file exists in the current directory
  // return nullptr if the file does not exist
  if (fs->cwd == -1)
  {
    return nullptr;
  }
  
  FCB dir = fs->start_of_fcb[fs->cwd];
  FCB *current = dir.dir_files;
  while (current != nullptr)
  {
    if (strmatch(current->filename, s))
    {
      // found
      return current;
    }
    current = current->next;
  }
  return nullptr;  // no such file in this directory
}

__device__ bool file_exists_in_curr_dir(FileSystem *fs, char* s) {
  return get_file_in_curr_dir(fs, s) != nullptr;
}

__device__ int get_fcb_idx(FileSystem *fs, FCB* fcb) {
  // get the index of the fcb in the fcb array
  return (int)(fcb - fs->start_of_fcb);
}

__device__ bool isdirname(char *s) {
  // check if a string is a directory name
  // a directory name ends with '/'
  int len = my_strlen(s);
  return s[len-2] == '/';
}

__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */
  // s ends with '\0'
  // op: open mode, G_READ or G_WRITE
  // returns the file pointer, which is the index of the FCB entry
  gtime++;
  bool file_exists = file_exists_in_curr_dir(fs, s);
  FCB * target_fcb = get_file_in_curr_dir(fs, s);
  if (isdirname(s))
  {
    file_exists = false;
  }
  

  switch (op)
  {
    case G_READ:
      // find file with the filename among all files, returns the index of the FCB
      if (file_exists) {
        return get_fcb_idx(fs, target_fcb);
      }
      assert(0);  // file not found
      break;
    case G_WRITE:
      // create the directory for the new file, if it's not already there; returns the address of the new FCB
      if (file_exists) {
        // have to empty the file in the next write operation
        // in which we will check the `size` attribute, if it's not 0, we will free the blocks
        int fcb_idx = get_fcb_idx(fs, target_fcb);
        set_gtime_recursive(fs, fcb_idx, gtime);
        printf("fs_open file %s exists, index %d\n", s, fcb_idx);
        return fcb_idx;
      } else {  // file not exists
        // allocate a new fcb index for the newly-created file
        for (int i = 0; i < fs->FCB_ENTRIES; i++)
        { // find an unused fcb
          target_fcb = fs->start_of_fcb + i;
          if (!target_fcb->is_on)
          {
            // mark the FCB as on and set its attrs
            target_fcb->size = 0;  // size at creation
            target_fcb->modified_time = gtime;
            target_fcb->creation_time = gtime;  // time at creation
            target_fcb->start_block_idx = 0;
            target_fcb->is_on = true;

            target_fcb->is_dir = false;
            target_fcb->parent_dir_idx = fs->start_of_fcb[fs->cwd].parent_dir_idx;
            target_fcb->dir_idx = fs->cwd;

            target_fcb->dir_files = nullptr;


            bool is_dir_name = isdirname(s);
            if (is_dir_name)
            {
              // this is a directory
              target_fcb->is_dir = true;
              target_fcb->parent_dir_idx = fs->cwd;
              target_fcb->dir_idx = i;
              s[my_strlen(s)-2] = '\0';  // remove the last '/'
              printf("fs_open dir %s created, index %d\n", s, i);
              printf("Parent dir idx %d\n", target_fcb->parent_dir_idx);
            }
            
            // copy the filename
            {
              int idx = 0;
              while (s[idx] != '\0')
              {
                fs->start_of_fcb[i].filename[idx] = s[idx];
                idx++;
              }
              fs->start_of_fcb[i].filename[idx] = '\0';
            }

            if (is_dir_name && (target_fcb->parent_dir_idx == -1))
            {
              // this is the root directory
              return target_fcb->dir_idx;
            }
            

            // append the new file to the current directory's doubly linked list
            {
              FCB *curr_dir = &fs->start_of_fcb[fs->cwd];
              if (curr_dir->dir_files == nullptr) {
                // the current directory is empty
                curr_dir->dir_files = target_fcb;
                target_fcb->prev = curr_dir;
                target_fcb->next = nullptr;
              } else {
                // the current directory is not empty
                FCB *last_file = curr_dir->dir_files;
                while (last_file->next != nullptr)
                {
                  last_file = last_file->next;
                }
                last_file->next = target_fcb;
                target_fcb->prev = last_file;
                target_fcb->next = nullptr;
              }
              
            }


            // add the filename to the directory file content
            {
              uchar * cwd_content = get_content(fs, fs->start_of_fcb[fs->cwd].start_block_idx, 0);
              int cwd_curr_size = fs->start_of_fcb[fs->cwd].size;
              uchar * input = new uchar[cwd_curr_size + my_strlen(s)];

              my_memcpy((char*)input, (char*)cwd_content, cwd_curr_size);
              my_memcpy((char*)(input+cwd_curr_size), (char*)s, my_strlen(s));
              
              printf("fs_open new fcb %s, index %d\n", s, i);
              
              fs_write(fs, input, cwd_curr_size + my_strlen(s), fs->cwd);

              delete[] input;
            }
            set_gtime_recursive(fs, i, gtime);

            return i;
          }
        }
        assert(0);  // no empty FCB
      }
      break;

    default:
      assert(0);  // no such option
      break;
  }
}

__device__ u32 block_of_bytes(FileSystem *fs, u32 bytes) {
  // returns how many blocks the `bytes` information will occupy
  u32 ret = bytes / fs->STORAGE_BLOCK_SIZE;
  if ((bytes % fs->STORAGE_BLOCK_SIZE) != 0) {
    ret++;
  }
  return ret;
}

__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
  // fp the index of the FCB
  assert(fs->start_of_fcb[fp].is_on);
  uchar *start = fs->start_of_contents + fs->start_of_fcb[fp].start_block_idx * fs->STORAGE_BLOCK_SIZE;
  FCB fcb = fs->start_of_fcb[fp];   // the fcb for this file

  printf("fs_read %d bytes from %s\n", size, fcb.filename);
  
  // read `size` bytes to buffer `output`
  for (u32 i = 0; i < size; i++)
  {
    output[i] = start[i];
  } 
}

__device__ void block_move(FileSystem *fs, int target_block_idx, int source_block_idx) {
  printf("moving block %d to %d\n", source_block_idx, target_block_idx);

  uchar *target_start = fs->start_of_contents + target_block_idx * fs->STORAGE_BLOCK_SIZE;
  uchar *source_start = fs->start_of_contents + source_block_idx * fs->STORAGE_BLOCK_SIZE;
  for (int i = 0; i < fs->STORAGE_BLOCK_SIZE; i++)
  {
    target_start[i] = source_start[i];
    source_start[i] = 0;
  }
  mark_block_unused(fs, source_block_idx);
  mark_block_used(fs, target_block_idx);
}

__device__ u16 alloc_new_blocks(FileSystem *fs, int target_block_size) {
  // allocate contiguous blocks with `target_block_size`, register it in the bitmap
  // return the index of the first block
  // if no enough contiguous blocks, have to manage the fragmentation
  printf("allocating %d blocks in alloc_new_blocks\n", target_block_size);
  int current_block_idx = 0;
  int block_count = 0;
  while (current_block_idx < fs->STORAGE_BLOCK_COUNT)
  {
    if (check_block_used(fs, current_block_idx)) {
      // this block is used, reset the counter
      block_count = 0;
    } else {
      block_count++;
      if (block_count == target_block_size) {
        // found enough contiguous blocks
        printf("contiguous block found, returning block %d, span %d\n", current_block_idx - target_block_size + 1, target_block_size);
        // mark blocks as used
        for (int i = 0; i < target_block_size; i++)
        {
          mark_block_used(fs, current_block_idx - target_block_size + 1 + i);
        }
        
        return current_block_idx - target_block_size + 1;
      }
    }
    current_block_idx++;
  }

  printf("No enough contiguous blocks, have to manage the fragmentation\n");
  // not enough contiguous space, have to manage the fragmentation
  // compation algorithm
  int first_unused_block_idx = 0;
  while (true)
  {
    // find the first unused block idx
    while (first_unused_block_idx < fs->STORAGE_BLOCK_COUNT)
    {
      if (!check_block_used(fs, first_unused_block_idx)) {
        break;
      }
      first_unused_block_idx++;
    }
    
    
    current_block_idx = first_unused_block_idx+1;
    // find the next used block idx
    while (current_block_idx < fs->STORAGE_BLOCK_COUNT)
    {
      if (check_block_used(fs, current_block_idx)) {
        break;
      }
      current_block_idx++;
    }

    if (current_block_idx >= fs->STORAGE_BLOCK_COUNT) {
      // no more used blocks
      break;
    }

    // swap the two blocks
    block_move(fs, first_unused_block_idx, current_block_idx);
    // reassign the associated fcbs of the moved block
    for (int i = 0; i < fs->FCB_ENTRIES; i++)
    {
      FCB *fcb = fs->start_of_fcb + i;
      if (fcb->start_block_idx == current_block_idx)
      {
        printf("reassigning fcb block %d to %d\n", fcb->start_block_idx, first_unused_block_idx);
        fcb->start_block_idx = first_unused_block_idx;
        break;
      }
    }
    
    first_unused_block_idx++;
    current_block_idx++;
    if (current_block_idx >= fs->STORAGE_BLOCK_COUNT) {
      // no more used blocks
      break;
    }
  }

  // reallocate
  printf("reallocating %d blocks in alloc_new_blocks\n", target_block_size);
  current_block_idx = 0;
  block_count = 0;
  while (current_block_idx < fs->STORAGE_BLOCK_COUNT)
  {
    if (check_block_used(fs, current_block_idx)) {
      // this block is used, reset the counter
      block_count = 0;
    } else {
      block_count++;
      if (block_count == target_block_size) {
        // found enough contiguous blocks
        printf("contiguous block found, returning block %d, span %d\n", current_block_idx - target_block_size + 1, target_block_size);
        // mark blocks as used
        for (int i = 0; i < target_block_size; i++)
        {
          mark_block_used(fs, current_block_idx - target_block_size + 1 + i);
        }
        
        return current_block_idx - target_block_size + 1;
      }
    }
    current_block_idx++;
  }
  assert(0);  // fail to reallocate

}



__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	// write bytes to the file
  // fp the index of the FCB
  gtime++;
  set_gtime_recursive(fs, fp, gtime);

  uchar *start = fs->start_of_contents + fs->start_of_fcb[fp].start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
  FCB *fcb = fs->start_of_fcb+fp;   // the fcb for this file
  u16 start_block_idx = fcb->start_block_idx;
  
  printf("fs_write %d bytes into %s\n", size, fcb->filename);
  // printf("start_block_idx %d\n", start_block_idx);
  // printf("fcb->size %d\n", fcb->size);
  // printf("check used: %d\n", check_block_used(fs, start_block_idx));

  // if the file already exists, we have to free the blocks 
  for (u32 i = 0; i < block_of_bytes(fs, fcb->size); i++)
  {
    mark_block_unused(fs, start_block_idx+i);
  }
  // empty the bytes, replace by 0
  for (u32 i = 0; i < fcb->size; i++)
  {
    start[i] = 0;
  }


  // begin writing to new file
  bool can_directly_write = true;
  for (u32 i = 0; i < block_of_bytes(fs, size); i++)
  {
    if (check_block_used(fs, start_block_idx+i))
    {
      can_directly_write = false;
      break;
    }
  }
  
  if (can_directly_write)
  {
    printf("directly writing %d blocks starting from block %d\n", block_of_bytes(fs, size), start_block_idx);
    // directly write to it
    for (u32 i = 0; i < size; i++)
    {
      start[i] = input[i];
    }
    for (u32 i = 0; i < block_of_bytes(fs, size); i++)
    {
      mark_block_used(fs, start_block_idx+i);
    }
    fcb->size = size;
    
    return size;
    
  } else {
    // cannot directly write, need to fix fragmentation, then directly write
    fcb->start_block_idx = alloc_new_blocks(fs, block_of_bytes(fs, size));
    printf("resetting start_block_idx to %d\n", fcb->start_block_idx);
    // perform write
    start = fs->start_of_contents + fcb->start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
    for (u32 i = 0; i < size; i++)
    {
      start[i] = input[i];
    }
    for (u32 i = 0; i < block_of_bytes(fs, size); i++)
    {
      mark_block_used(fs, fcb->start_block_idx+i);
    }

    fcb->size = size;
    return size;
  }
}

__device__ void pwd_helper(FileSystem *fs, int fcb_idx) {
  // to print cwd, this should be called `pwd_helper(fs, fs->cwd)`

  // recursively print the path of the current directory
  FCB *fcb = fs->start_of_fcb + fcb_idx;
  if (fcb->parent_dir_idx == -1) {
    // root directory
    return;
  }
  pwd_helper(fs, fcb->parent_dir_idx);
  printf("/%s", fcb->filename);
}


// ls_d, ls_s, cd_p, pwd goes here
__device__ void fs_gsys(FileSystem *fs, int op)
{
  FCB cwd_fcb = fs->start_of_fcb[fs->cwd];
  uchar *cwd_content = get_content(fs, cwd_fcb.start_block_idx, 0);
  // get the number of files and subdirectories **in the current directory**
  int file_count = 0;
  for (int i = 0; i < cwd_fcb.size; i++)
  {
    if (cwd_content[i] == '\0')
    {
      file_count++;
    }
  }
  
  printf("number of files or dirs: %d\n", file_count);

	/* Implement ls_d, ls_s, cd_p, pwd operation here */
  switch (op)
  {
  case PWD:
  {
    printf("printing pwd\n");
    // printf("fs.cwd: %d\n", fs->cwd);
    if (cwd_fcb.parent_dir_idx == -1) {
      // root directory
      printf("/\n");
      break;
    } else {
      pwd_helper(fs, fs->cwd);
      printf("\n");
      break;
    }
  }
  case CD_P:
  {
    // cd to parent dir
    if (cwd_fcb.parent_dir_idx == -1) {
      // root directory
      printf("already in root directory\n");
      break;
    } else {
      printf("changed directory to parent directory\n");
      fs->cwd = cwd_fcb.parent_dir_idx;
      break;
    }
  }
  case LS_D:
  {

    // list the file name and order by modified time of files

    printf("===sort by modified time===\n");

    int last_item_time = (1<<15); // trace the time of last printed file
    // print the most recent modified file before the last item
    for (int i = 0; i < file_count; i++)  // print each file once
    {
      int latest_modified_time = 0;
      FCB latest_fcb;

      // tokenize the file name in the cwd content
      uchar current_byte;
      int token_start_idx = 0;

      // find the most recently modified file or subdir
      for (int j = 0; j < cwd_fcb.size; j++)
      {
        current_byte = *get_content(fs, cwd_fcb.start_block_idx, j);
        if (current_byte == '\0')
        {
          // get this full token
          char token[21];
          my_memcpy(token, (char*)get_content(fs, cwd_fcb.start_block_idx, token_start_idx), 21);
          // get the fcb
          FCB *fcb = get_file_in_curr_dir(fs, token);
          if (fcb->is_on && (fcb->modified_time > latest_modified_time) && (fcb->modified_time < last_item_time))
          {
            latest_fcb = *fcb;
            latest_modified_time = fcb->modified_time;
          }
          
          token_start_idx = j+1;
        }
        
      }

      last_item_time = latest_fcb.modified_time;
      if (latest_fcb.is_dir)
      {
        printf("%s d\n", latest_fcb.filename);
      } else {
        printf("%s\n", latest_fcb.filename);
      }
      
      // printf("%s   time%d\n", latest_fcb.filename, last_item_time);
    }
    break;
  }

  case LS_S:
  {
      
    printf("===sort by file size===\n");
    // If there are several files with the same size, then first create first print.

    
    u32 last_item_size = (1<<31); // the distinct size of the last printed file
    int print_count = 0;

    while (print_count < file_count)
    {

      int largest_file_size = -1;
      // get the largest file size less than `last_item_size`
      FCB * curr = cwd_fcb.dir_files;
      while (curr != nullptr)
      {
        if (curr->is_on && (curr->size < last_item_size) && ((int)(curr->size) > largest_file_size))
        {
          largest_file_size = curr->size;
        }
        curr = curr->next;
      }
      
      // print all files with the same size
      last_item_size = largest_file_size;

      // printf("largest file size: %d\n", largest_file_size);

      // count the number of files with the size of largest_file_size

      int largest_file_count = 0;

      curr = cwd_fcb.dir_files;
      while (curr != nullptr)
      {
        if (curr->is_on && (curr->size == largest_file_size))
        {
          largest_file_count++;
        }
        curr = curr->next;
      }

      // printf("largest_file_count: %d\n", largest_file_count);

      // find the file with the file size of largest_file_size and the earliest created time among all unprinted items
      u16 last_item_time = 0;
      while (largest_file_count > 0)
      {
        u16 earliest_created_time = (1<<15);
        FCB *earliest_fcb;
        curr = cwd_fcb.dir_files;
        
        while (curr != nullptr)
        {
          assert(curr->is_on);
          if ((curr->size == largest_file_size) && (curr->creation_time < earliest_created_time) && (curr->creation_time > last_item_time))
          {
            earliest_created_time = curr->creation_time;
            earliest_fcb = curr;
            // printf("earliest_created_time: %d\n", curr->creation_time);
          }
          curr = curr->next;
        }
        last_item_time = earliest_created_time;
        largest_file_count--;
        print_count++;
        if (earliest_fcb->is_dir)
        {
          printf("%s %d d\n", earliest_fcb->filename, earliest_fcb->size);
        } else {
          printf("%s %d\n", earliest_fcb->filename, earliest_fcb->size);
        }
        
      }
    }
    break;
  }
  default:
    assert(0);
    break;  // no such option
  } // end of switch
  
}


// rm, cd, mkdir, rm_rf goes here
__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
  // find the specific file in the FCB
  bool file_exists = file_exists_in_curr_dir(fs, s);
  FCB *target_fcb = get_file_in_curr_dir(fs, s);
  

	/* Implement rm operation here */
  if (op == RM)
  {
    // delete the specific file
    if (!file_exists)
    {
      assert(0);  // file not found
    } else {
      target_fcb->is_on = false;

      // remove the item in the directory's linked list
      assert(target_fcb->prev != nullptr);
      target_fcb->prev->next = target_fcb->next;
      if (target_fcb->prev->is_dir)
      {
        target_fcb->prev->dir_files = target_fcb->next;
      }
      
      if (target_fcb->next != nullptr)
      {
        target_fcb->next->prev = target_fcb->prev;
      }
           
      // remove the item in the parent dir's content
      {
        printf("removing %s in dir %s\n", target_fcb->filename, fs->start_of_fcb[fs->cwd].filename);

        uchar * cwd_content = get_content(fs, fs->start_of_fcb[fs->cwd].start_block_idx, 0);
        int cwd_curr_size = fs->start_of_fcb[fs->cwd].size;
        
        uchar *new_input = new uchar[cwd_curr_size];
        my_memcpy((char*)new_input, (char*)cwd_content, cwd_curr_size);

        // find the position of this filename
        uchar current_byte;
        int token_start_idx = 0;
        for (int j = 0; j < cwd_curr_size; j++)
        {
          current_byte = new_input[j];
          // printf("I am examining byte %c\n", current_byte);
          if (current_byte == '\0')
          {
            // printf("token starts with %c\n", *(new_input+token_start_idx));
            if (strmatch(target_fcb->filename, (char*)(new_input+token_start_idx)))
            {
              // printf("match\n");
              my_memcpy((char*)(new_input+token_start_idx), (char*)(new_input+j+1), my_strlen(target_fcb->filename));
              break;
            }
            token_start_idx = j+1;
          }
        }

        fs_write(fs, new_input, cwd_curr_size-my_strlen(target_fcb->filename), fs->cwd);
        delete[] new_input;
      }

      // free the content memory
      uchar *start = get_content(fs, target_fcb->start_block_idx, 0); // the initial byte of the file content
      printf("fs_delete removing %d bytes of %s, start from block %d span %d\n", target_fcb->size, target_fcb->filename, target_fcb->start_block_idx, block_of_bytes(fs, target_fcb->size));

      // free the blocks  
      for (u32 i = 0; i < block_of_bytes(fs, target_fcb->size); i++)
      {
        mark_block_unused(fs, target_fcb->start_block_idx+i);
      }
      // empty the bytes, replace by 0
      for (u32 i = 0; i < target_fcb->size; i++)
      {
        start[i] = 0;
      }
    }
    
  } else if (op == MKDIR)
  {
    // create a new directory
    if (file_exists) {
      assert(0);  // directory already exists
    } else {
      // append '/' at the end of s
      int len = my_strlen(s);
      char tmp[21];
      my_memcpy(tmp, s, len);
      tmp[len-1] = '/';
      tmp[len] = '\0';
      u32 fp = fs_open(fs, tmp, G_WRITE);
    }
  } else if (op == CD) {
    assert(file_exists);  // if assertion failed, the directory does not exist
    fs->cwd = target_fcb->dir_idx;
    printf("change directory to %s, index %d\n", s, fs->cwd);
  } else if (op == RM_RF) {
    // Remove the app directory and all its subdirectories and files recursively
    assert(file_exists);  // if assertion failed, the directory or file does not exist
    printf("RM_RF called, to remove %s\n", s);
    
    // locate the additional fcb
    if (!target_fcb->is_dir)
    {
      // remove that regular file only
      printf("rm_rf removing regular file %s\n", s);
      fs_gsys(fs, RM, s);
    } else {
      // recursively remove the directory, its subdirectories and files

      // first CD into the target dir, after it has been removed CD back
      fs_gsys(fs, CD, s);

      FCB *cwd_fcb = fs->start_of_fcb + fs->cwd;
      FCB *curr = cwd_fcb->dir_files;
      // move to the last one and move forward
      while (curr->next != nullptr)
      {
        curr = curr->next;
      }
      while (curr != cwd_fcb)
      {
        FCB *previous = curr->prev;
        if (!curr->is_dir)
        {
          printf("rm_rf removing regular file %s\n", curr->filename);
          fs_gsys(fs, RM, curr->filename);
        } else {
          printf("rm_rf removing dir %s\n", curr->filename);
          fs_gsys(fs, RM_RF, curr->filename);
        }
        curr = previous;
      }
      // CD back
      fs_gsys(fs, CD_P);

      // since we have emptied the dir's subunits, we can now remove it as a regular file
      fs_gsys(fs, RM, target_fcb->filename);

    }
    
  }
  
  
}
