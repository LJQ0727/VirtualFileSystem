﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS,
              FCB_additional *start_of_fcb_additional)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
  fs->STORAGE_BLOCK_COUNT = (fs->STORAGE_SIZE - fs->SUPERBLOCK_SIZE - (fs->FCB_SIZE * fs->FCB_ENTRIES)) / fs->STORAGE_BLOCK_SIZE;

  // init some custom pointers
  fs->start_of_fcb = (FCB*)(volume+SUPERBLOCK_SIZE);
  fs->start_of_superblock = volume;
  fs->start_of_contents = volume + FILE_BASE_ADDRESS;
  fs->start_of_fcb_additional = start_of_fcb_additional;
  fs->cwd = -1;   // the root directory will have parent dir index -1

  // initialize volume
  for (int i = 0; i < VOLUME_SIZE; i++)
    volume[i] = 0;

  // initialize fcb_additional
  for (int i = 0; i < FCB_ENTRIES; i++) {
    start_of_fcb_additional[i].is_dir = false;
    start_of_fcb_additional[i].number_of_files = 0;
    start_of_fcb_additional[i].parent_dir_idx = -1;
  }

  // make root directory and cd to it
  fs_gsys(fs, MKDIR, "/\0");
  fs_gsys(fs, CD, "/\0");
    
}


__device__ bool strmatch(char *start1, char* start2) {
  // match two strings, return true if they are the same
  int idx = 0;
  while (!(start1[idx] == '\0' || start2[idx] == '\0'))
  {
    if (start1[idx] != start2[idx])
    {
      return false;
    }
    idx++;
  }
  if (start1[idx] != start2[idx])
  {
    return false;
  }

  return true;
}

__device__ void mark_block_used(FileSystem *fs, int block_idx) {
  // mark a block as used in the superblock
  // operate on only one block at a time
  uchar bitmap = fs->start_of_superblock[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  fs->start_of_superblock[block_idx/8] = bitmap | mask;
}

__device__ void mark_block_unused(FileSystem *fs, int block_idx) {
  // mark a block as unused in the superblock
  // operate on only one block at a time
  uchar bitmap = fs->start_of_superblock[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  fs->start_of_superblock[block_idx/8] = bitmap & ~mask;
}

__device__ bool check_block_used(FileSystem *fs, int block_idx) {
  uchar bitmap = fs->start_of_superblock[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  return bitmap & mask;
}


__device__ void my_memcpy(char *destination, char *source, int size) {
  // this will not automatically add '\0' to the string
  for (int i = 0; i < size; i++)
  {
    destination[i] = source[i];
  }
  
}


__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */
  // s ends with '\0'
  // op: open mode, G_READ or G_WRITE
  // returns the file pointer, which is the index of the FCB entry
  gtime++;

  // find if the specific file already exists in the FCB
  bool file_exists = false;
  int fcb_idx = 0;
  for (int i = 0; i < fs->FCB_ENTRIES; i++)
  {
    FCB target_fcb = fs->start_of_fcb[i];
    if (target_fcb.is_on && strmatch(target_fcb.filename, s))
    {
      file_exists = true;
      fcb_idx = i;
      break;
    }
  }
  
  switch (op)
  {
    case G_READ:
      // find file with the filename among all files, returns the index of the FCB
      if (file_exists) {
        return fcb_idx;
      }
      assert(0);  // file not found
      break;
    case G_WRITE:
      // create the directory for the new file, if it's not already there; returns the address of the new FCB
      if (file_exists) {
        // have to empty the file in the next write operation
        // in which we will check the `size` attribute, if it's not 0, we will free the blocks
        printf("fs_open file %s exists, index %d\n", s, fcb_idx);
        return fcb_idx;
      } else {  // file not exists
        // allocate a new fcb index for the newly-created file
        for (int i = 0; i < fs->FCB_ENTRIES; i++)
        { // find an unused fcb
          FCB target_fcb = fs->start_of_fcb[i];
          if (!target_fcb.is_on)
          {
            // mark the FCB as on
            fs->start_of_fcb[i].is_on = true;
            fs->start_of_fcb[i].modified_time = gtime;
            fs->start_of_fcb[i].size = 0;  // size at creation
            fs->start_of_fcb[i].creation_time = gtime;  // time at creation
            fs->start_of_fcb[i].start_block_idx = 0;
            // copy the filename
            int idx = 0;
            while (s[idx] != '\0')
            {
              fs->start_of_fcb[i].filename[idx] = s[idx];
              idx++;
            }
            fs->start_of_fcb[i].filename[idx] = '\0';

            fs->start_of_fcb[i].size = 0; // no content for this file for now
            
            printf("fs_open new fcb %s, index %d\n", s, i);
            return i;
          }
        }
        assert(0);  // no empty FCB
      }
      break;

    default:
      assert(0);  // no such option
      break;
    return fcb_idx;
  }
}

__device__ u32 block_of_bytes(FileSystem *fs, u32 bytes) {
  // returns how many blocks the `bytes` information will occupy
  u32 ret = bytes / fs->STORAGE_BLOCK_SIZE;
  if ((bytes % fs->STORAGE_BLOCK_SIZE) != 0) {
    ret++;
  }
  return ret;
}

__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
  // fp the index of the FCB
  assert(fs->start_of_fcb[fp].is_on);
  uchar *start = fs->start_of_contents + fs->start_of_fcb[fp].start_block_idx * fs->STORAGE_BLOCK_SIZE;
  FCB fcb = fs->start_of_fcb[fp];   // the fcb for this file

  printf("fs_read %d bytes from %s\n", size, fcb.filename);
  
  // read `size` bytes to buffer `output`
  for (u32 i = 0; i < size; i++)
  {
    output[i] = start[i];
  } 
}

__device__ void block_move(FileSystem *fs, int target_block_idx, int source_block_idx) {
  printf("moving block %d to %d\n", source_block_idx, target_block_idx);

  uchar *target_start = fs->start_of_contents + target_block_idx * fs->STORAGE_BLOCK_SIZE;
  uchar *source_start = fs->start_of_contents + source_block_idx * fs->STORAGE_BLOCK_SIZE;
  for (int i = 0; i < fs->STORAGE_BLOCK_SIZE; i++)
  {
    target_start[i] = source_start[i];
    source_start[i] = 0;
  }
  mark_block_unused(fs, source_block_idx);
  mark_block_used(fs, target_block_idx);
}

__device__ u16 alloc_new_blocks(FileSystem *fs, int target_block_size) {
  // allocate contiguous blocks with `target_block_size`, register it in the bitmap
  // return the index of the first block
  // if no enough contiguous blocks, have to manage the fragmentation
  printf("allocating %d blocks in alloc_new_blocks\n", target_block_size);
  int current_block_idx = 0;
  int block_count = 0;
  while (current_block_idx < fs->STORAGE_BLOCK_COUNT)
  {
    if (check_block_used(fs, current_block_idx)) {
      // this block is used, reset the counter
      block_count = 0;
    } else {
      block_count++;
      if (block_count == target_block_size) {
        // found enough contiguous blocks
        printf("contiguous block found, returning block %d, span%d\n", current_block_idx - target_block_size + 1, target_block_size);
        // mark blocks as used
        for (int i = 0; i < target_block_size; i++)
        {
          mark_block_used(fs, current_block_idx - target_block_size + 1 + i);
        }
        
        return current_block_idx - target_block_size + 1;
      }
    }
    current_block_idx++;
  }

  printf("No enough contiguous blocks, have to manage the fragmentation\n");
  // not enough contiguous space, have to manage the fragmentation
  // compation algorithm
  int first_unused_block_idx = 0;
  while (true)
  {
    // find the first unused block idx
    while (first_unused_block_idx < fs->STORAGE_BLOCK_COUNT)
    {
      if (!check_block_used(fs, first_unused_block_idx)) {
        break;
      }
      first_unused_block_idx++;
    }
    
    
    current_block_idx = first_unused_block_idx+1;
    // find the next used block idx
    while (current_block_idx < fs->STORAGE_BLOCK_COUNT)
    {
      if (check_block_used(fs, current_block_idx)) {
        break;
      }
      current_block_idx++;
    }

    if (current_block_idx >= fs->STORAGE_BLOCK_COUNT) {
      // no more used blocks
      break;
    }

    // swap the two blocks
    block_move(fs, first_unused_block_idx, current_block_idx);
    // reassign the associated fcbs of the moved block
    for (int i = 0; i < fs->FCB_ENTRIES; i++)
    {
      FCB *fcb = fs->start_of_fcb + i;
      if (fcb->start_block_idx == current_block_idx)
      {
        printf("reassigning fcb block %d to %d\n", fcb->start_block_idx, first_unused_block_idx);
        fcb->start_block_idx = first_unused_block_idx;
        break;
      }
    }
    
    first_unused_block_idx++;
    current_block_idx++;
    if (current_block_idx >= fs->STORAGE_BLOCK_COUNT) {
      // no more used blocks
      break;
    }
  }

  // reallocate
  printf("reallocating %d blocks in alloc_new_blocks\n", target_block_size);
  current_block_idx = 0;
  block_count = 0;
  while (current_block_idx < fs->STORAGE_BLOCK_COUNT)
  {
    if (check_block_used(fs, current_block_idx)) {
      // this block is used, reset the counter
      block_count = 0;
    } else {
      block_count++;
      if (block_count == target_block_size) {
        // found enough contiguous blocks
        printf("contiguous block found, returning block %d, span%d\n", current_block_idx - target_block_size + 1, target_block_size);
        // mark blocks as used
        for (int i = 0; i < target_block_size; i++)
        {
          mark_block_used(fs, current_block_idx - target_block_size + 1 + i);
        }
        
        return current_block_idx - target_block_size + 1;
      }
    }
    current_block_idx++;
  }
  assert(0);  // fail to reallocate

}

__device__ FCB_additional * get_fcb_additional(FileSystem *fs, int fcb_idx) {
  // get the additional fcb information pointer
  return fs->start_of_fcb_additional + fcb_idx;
}

__device__ uchar * get_content(FileSystem *fs, int block_idx, int byte_offset) {
  // given a block index, get the pointer to the content of a file (or directory)
  return fs->start_of_contents + block_idx * fs->STORAGE_BLOCK_SIZE + byte_offset;
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	/* Implement write operation here */
  // fp the index of the FCB
  gtime++;


  uchar *start = fs->start_of_contents + fs->start_of_fcb[fp].start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
  FCB *fcb = fs->start_of_fcb+fp;   // the fcb for this file
  u16 start_block_idx = fcb->start_block_idx;
  
  printf("fs_write %d bytes into %s\n", size, fcb->filename);
  // printf("start_block_idx %d\n", start_block_idx);
  // printf("fcb->size %d\n", fcb->size);
  // printf("check used: %d\n", check_block_used(fs, start_block_idx));

  // if the file already exists, we have to free the blocks 
  for (u32 i = 0; i < block_of_bytes(fs, fcb->size); i++)
  {
    mark_block_unused(fs, start_block_idx+i);
  }
  // empty the bytes, replace by 0
  for (u32 i = 0; i < fcb->size; i++)
  {
    start[i] = 0;
  }


  // begin writing to new file
  bool can_directly_write = true;
  for (u32 i = 0; i < block_of_bytes(fs, size); i++)
  {
    if (check_block_used(fs, start_block_idx+i))
    {
      can_directly_write = false;
      break;
    }
  }
  
  if (can_directly_write)
  {
    printf("directly writing %d blocks starting from block %d\n", block_of_bytes(fs, size), start_block_idx);
    // directly write to it
    for (u32 i = 0; i < size; i++)
    {
      start[i] = input[i];
    }
    for (u32 i = 0; i < block_of_bytes(fs, size); i++)
    {
      mark_block_used(fs, start_block_idx+i);
    }
    fcb->size = size;
    fcb->modified_time = gtime;
    // printf("modified time of %s: %d\n", fcb->filename ,gtime);
    
    return size;
    
  } else {
    // cannot directly write, need to fix fragmentation, then directly write
    fcb->start_block_idx = alloc_new_blocks(fs, block_of_bytes(fs, size));
    printf("cannot directly write, resetting start_block_idx to %d\n", fcb->start_block_idx);
    // perform write
    start = fs->start_of_contents + fcb->start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
    for (u32 i = 0; i < size; i++)
    {
      start[i] = input[i];
    }
    for (u32 i = 0; i < block_of_bytes(fs, size); i++)
    {
      mark_block_used(fs, fcb->start_block_idx+i);
    }

    fcb->size = size;
    fcb->modified_time = gtime;
    return size;
  }
  
  
  
  
}

__device__ void pwd_helper(FileSystem *fs, int fcb_idx) {
  // to print cwd, this should be called `pwd_helper(fs, fs->cwd)`

  // recursively print the path of the current directory
  FCB *fcb = fs->start_of_fcb + fcb_idx;
  FCB_additional *fcb_additional = get_fcb_additional(fs, fcb_idx);
  if (fcb_additional->parent_dir_idx == -1) {
    // root directory
    return;
  }
  pwd_helper(fs, fcb_additional->parent_dir_idx);
  printf("/%s", fcb->filename);
}

__device__ int get_fcb_by_name(FileSystem *fs, char *name) {
  // get the fcb index by name
  for (int i = 0; i < fs->FCB_ENTRIES; i++)
  {
    FCB *fcb = fs->start_of_fcb + i;
    if (fcb->is_on && strmatch(fcb->filename, name)) {
      return i;
    }
  }
  assert(0);  // no such named fcb
  return -1;
}

// ls_d, ls_s, cd_p, pwd goes here
__device__ void fs_gsys(FileSystem *fs, int op)
{
  FCB cwd_fcb = fs->start_of_fcb[fs->cwd];
  uchar *cwd_content = get_content(fs, cwd_fcb.start_block_idx, 0);
  // count number of files and subdirectories **in the current directory**
  int file_count = 0;
  for (int i = 0; i < cwd_fcb.size; i++)
  {
    if (cwd_content[i] == '\0')
    {
      file_count++;
    }
  }
  
  // printf("number of files or dirs: %d\n", file_count);

	/* Implement ls_d, ls_s, cd_p, pwd operation here */
  switch (op)
  {
  case PWD:
  {
    pwd_helper(fs, fs->cwd);
    printf("\n");
    break;
  }
  case LS_D:
  {

    // list the file name and order by modified time of files

    printf("===sort by modified time===\n");

    int last_item_time = (1<<15); // trace the time of last printed file
    // print the most recent modified file before the last item
    for (int i = 0; i < file_count; i++)  // print each file once
    {
      int latest_modified_time = 0;
      FCB latest_fcb;
      int latest_fcb_idx = -1;

      // tokenize the file name in the cwd content
      uchar current_byte;
      int token_start_idx = 0;

      
      // find the most recently modified file or subdir
      for (int j = 0; j < cwd_fcb.size; j++)
      {
        current_byte = *get_content(fs, cwd_fcb.start_block_idx, j);
        if (current_byte == '\0')
        {
          // get this full token
          char token[21];
          my_memcpy(token, (char*)get_content(fs, cwd_fcb.start_block_idx, token_start_idx), 21);
          // get the fcb
          int fcb_idx = get_fcb_by_name(fs, token);
          FCB *fcb = fs->start_of_fcb + fcb_idx;
          if (fcb->is_on && (fcb->modified_time > latest_modified_time) && (fcb->modified_time < last_item_time))
          {
            latest_fcb = *fcb;
            latest_modified_time = fcb->modified_time;
            latest_fcb_idx = fcb_idx;
          }
          
          token_start_idx = j+1;
        }
        
      }

      FCB_additional *fcb_additional = get_fcb_additional(fs, latest_fcb_idx);
      last_item_time = latest_fcb.modified_time;
      if (fcb_additional->is_dir)
      {
        printf("%s d\n", latest_fcb.filename);
      } else {
        printf("%s\n", latest_fcb.filename);
      }
      
      // printf("%s   time%d\n", latest_fcb.filename, last_item_time);
    }
    

    break;
  }


  case LS_S:
  {
      
    printf("===sort by file size===\n");
    // If there are several files with the same size, then first create first print.

    
    u16 last_item_size = (1<<15); // the distinct size of the last printed file
    int print_count = 0;

    while (print_count < file_count)
    {
      int largest_file_size = 0;

      // tokenize the file name in the cwd content
      uchar current_byte;
      int token_start_idx = 0;

      
      // find the largest file or subdir
      for (int j = 0; j < cwd_fcb.size; j++)
      {
        current_byte = *get_content(fs, cwd_fcb.start_block_idx, j);
        if (current_byte == '\0')
        {
          // get this full token
          char token[21];
          my_memcpy(token, (char*)get_content(fs, cwd_fcb.start_block_idx, token_start_idx), 21);
          // get the fcb
          int fcb_idx = get_fcb_by_name(fs, token);
          FCB *fcb = fs->start_of_fcb + fcb_idx;
          if (fcb->is_on && (fcb->size > largest_file_size) && (fcb->modified_time < last_item_size))
          {
            largest_file_size = fcb->size;
          }
          
          token_start_idx = j+1;
        }
        
      }
      last_item_size = largest_file_size;

      // printf("last item size: %d\n", last_item_size);
      // printf("largest file size: %d\n", largest_file_size);

      // count the number of files with the size of largest_file_size
      token_start_idx = 0;
      int largest_file_count = 0;
      // find the file or subdir with the same size
      for (int j = 0; j < cwd_fcb.size; j++)
      {
        current_byte = *get_content(fs, cwd_fcb.start_block_idx, j);
        if (current_byte == '\0')
        {
          // get this full token
          char token[21];
          my_memcpy(token, (char*)get_content(fs, cwd_fcb.start_block_idx, token_start_idx), 21);
          // get the fcb
          int fcb_idx = get_fcb_by_name(fs, token);
          FCB *fcb = fs->start_of_fcb + fcb_idx;
          if (fcb->is_on && (fcb->size == largest_file_size))
          {
            largest_file_count++;
          }
          
          token_start_idx = j+1;
        }
        
      }
      // printf("largest file size: %d, count: %d\n", largest_file_size, largest_file_count);

      u16 last_item_time = 0;
      for (int i = 0; i < largest_file_count; i++)
      {
        // find the file with the file size of largest_file_size and the earliest created time among all unprinted items

        u16 earliest_created_time = (1<<15);
        FCB earliest_fcb;
        int earliest_fcb_idx;

        for (int j = 0; j < cwd_fcb.size; j++)
        {
          current_byte = *get_content(fs, cwd_fcb.start_block_idx, j);
          if (current_byte == '\0')
          {
            // get this full token
            char token[21];
            my_memcpy(token, (char*)get_content(fs, cwd_fcb.start_block_idx, token_start_idx), 21);
            // get the fcb
            int fcb_idx = get_fcb_by_name(fs, token);
            FCB *fcb = fs->start_of_fcb + fcb_idx;
            if (fcb->is_on && (fcb->size == largest_file_size) && (fcb->creation_time < earliest_created_time) && (fcb->creation_time > last_item_time))
            {
              earliest_fcb = *fcb;
              earliest_created_time = fcb->creation_time;
              earliest_fcb_idx = fcb_idx;
            }
            
            token_start_idx = j+1;
          }
        }
        last_item_time = earliest_fcb.creation_time;
        FCB_additional * additional = get_fcb_additional(fs, earliest_fcb_idx);
        if (additional->is_dir)
        {
          printf("%s %d d\n", earliest_fcb.filename, earliest_fcb.size);
        } else {
          printf("%s %d\n", earliest_fcb.filename, earliest_fcb.size);
        }
      }
      print_count += largest_file_count;
    }
    break;
  }
  default:
    break;  // no such option
  } // end of switch
  
}


// rm, cd, mkdir, rm_rf goes here
__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
  // find the specific file in the FCB
  bool file_exists = false;
  int fcb_idx = 0;
  FCB *target_fcb;

  for (int i = 0; i < fs->FCB_SIZE; i++)
  {
    target_fcb = &fs->start_of_fcb[i];
    if (target_fcb->is_on && strmatch(target_fcb->filename, s))
    {
      file_exists = true;
      fcb_idx = i;
      break;
    }
  }

	/* Implement rm operation here */
  if (op == RM)
  {
    // delete the specific file
    if (!file_exists)
    {
      assert(0);  // file not found
    } else {
      target_fcb->is_on = false;

      // free the content memory
      uchar *start = fs->start_of_contents + target_fcb->start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
      
      printf("fs_delete removing %d bytes of %s, start from block %d span %d\n", target_fcb->size, target_fcb->filename, target_fcb->start_block_idx, block_of_bytes(fs, target_fcb->size));

      // free the blocks  
      for (u32 i = 0; i < block_of_bytes(fs, target_fcb->size); i++)
      {
        mark_block_unused(fs, target_fcb->start_block_idx+i);
      }
      // empty the bytes, replace by 0
      for (u32 i = 0; i < target_fcb->size; i++)
      {
        start[i] = 0;
      }
    }
    
  } else if (op == MKDIR)
  {
    // create a new directory
    if (file_exists) {
      assert(0);  // directory already exists
    } else {
      // find an empty FCB
      for (int i = 0; i < fs->FCB_SIZE; i++)
      {
        target_fcb = &fs->start_of_fcb[i];
        if (!target_fcb->is_on)
        {
          // create a new empty directory in this file
          FCB_additional * target_additional_fcb = get_fcb_additional(fs, i);
          target_fcb->is_on = true;
          target_fcb->size = 0;
          target_fcb->start_block_idx = 0;
          target_fcb->creation_time = gtime;
          target_fcb->modified_time = gtime;

          target_additional_fcb->is_dir = true;
          target_additional_fcb->number_of_files = 0;
          target_additional_fcb->parent_dir_idx = fs->cwd;  // set the parent directory to the current working directory

          // copy the dirname as filename
          int idx = 0;
          while (s[idx] != '\0')
          {
            fs->start_of_fcb[i].filename[idx] = s[idx];
            idx++;
          }
          fs->start_of_fcb[i].filename[idx] = '\0';

          printf("create new directory fcb %s, index %d\n", s, i);
          break;
        }
        assert(0);  // no empty FCB
      }
    }
  } else if (op == CD) {
    assert(file_exists);  // if assertion failed, the directory does not exist
    fs->cwd = fcb_idx;
  } else if (op == RM_RF) {
    // Remove the app directory and all its subdirectories and files recursively
    assert(file_exists);  // if assertion failed, the directory or file does not exist

    // locate the additional fcb
    FCB_additional * additional_fcb = get_fcb_additional(fs, fcb_idx);
    if (!additional_fcb->is_dir)
    {
      // remove that file only
      fs_gsys(fs, RM, s);
    } else {
      // recursively remove the directory, its subdirectories and files
      // read in the contained file or directory info
      // by tokenizing

      uchar current_byte;
      int token_start_idx = 0;

      for (int i = 0; i < target_fcb->size; i++)
      {
        current_byte = *get_content(fs, target_fcb->start_block_idx, i);
        if (current_byte == '\0')
        {
          // get this full token
          char token[21];
          my_memcpy(token, (char*)get_content(fs, target_fcb->start_block_idx, token_start_idx), i-token_start_idx+1);
          fs_gsys(fs, RM_RF, token);  // recursive call to remove subdir or subfile
          token_start_idx = i+1;
        }
        
      }
      // after all subdir and subfile are removed, remove myself
      target_fcb->is_on = false;

      // free the content memory
      uchar *start = fs->start_of_contents + target_fcb->start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
      
      printf("fs_rmrf removing %d bytes of directory %s, start from block %d span %d\n", target_fcb->size, target_fcb->filename, target_fcb->start_block_idx, block_of_bytes(fs, target_fcb->size));

      // free the blocks  
      for (u32 i = 0; i < block_of_bytes(fs, target_fcb->size); i++)
      {
        mark_block_unused(fs, target_fcb->start_block_idx+i);
      }
      // empty the bytes, replace by 0
      for (u32 i = 0; i < target_fcb->size; i++)
      {
        start[i] = 0;
      }
    }
    
  }
  
  
}
