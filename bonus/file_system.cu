﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>


#define START_OF_SUPERBLOCK (fs->volume)
#define START_OF_FCB ((FCB*)(fs->volume + fs->SUPERBLOCK_SIZE))
#define START_OF_CONTENTS (fs->volume + fs->FILE_BASE_ADDRESS)
#define STORAGE_BLOCK_COUNT ((fs->STORAGE_SIZE - fs->SUPERBLOCK_SIZE - (fs->FCB_SIZE * fs->FCB_ENTRIES)) / fs->STORAGE_BLOCK_SIZE)

typedef uint16_t u16;
typedef unsigned char uchar;
typedef uint32_t u32;

__device__ __managed__ u32 gtime = 0;

__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

  fs->cwd = -1;   // the root directory will have parent dir index -1

  // initialize volume
  for (int i = 0; i < VOLUME_SIZE; i++)
    volume[i] = 0;

  // make root directory and cd to it
  // append '/' at the end of s, indicating dir
  char tmp[3] = {'/', '/', '\0'};
  u32 fp = fs_open(fs, tmp, G_WRITE);
  fs->cwd = 0;  // root will be created at idx 0
}

__device__ bool strmatch(char *start1, char* start2);

__device__ void mark_fcb_on(FCB *fcb)
{
  fcb->size |= (1 << 31);
}

__device__ void mark_fcb_off(FCB *fcb)
{
  fcb->size &= ~(1 << 31);
}

__device__ bool check_fcb_on(FCB *fcb)
{
  // the fcb is on if its MSB of size is 1
  return fcb->size & (1<<31);
}

__device__ void mark_is_dir(FCB *fcb) {
  fcb->size |= (1 << 30);
}

__device__ void mark_not_dir(FCB *fcb) {
  fcb->size &= ~(1 << 30);
}

__device__ bool check_is_dir(FCB *fcb) {
  return fcb->size & (1<<30);
}

__device__ void mark_block_used(FileSystem *fs, int block_idx) {
  // mark a block as used in the superblock
  // operate on only one block at a time
  uchar bitmap = START_OF_SUPERBLOCK[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  START_OF_SUPERBLOCK[block_idx/8] = bitmap | mask;
}

__device__ void mark_block_unused(FileSystem *fs, int block_idx) {
  // mark a block as unused in the superblock
  // operate on only one block at a time
  uchar bitmap = START_OF_SUPERBLOCK[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  START_OF_SUPERBLOCK[block_idx/8] = bitmap & ~mask;
}

__device__ bool check_block_used(FileSystem *fs, int block_idx) {
  uchar bitmap = START_OF_SUPERBLOCK[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  return bitmap & mask;
}

__device__ void set_size_of_fcb(FCB *fcb, int size) {
  // Because we use the 2 MSB of `size` for compact storage, so need to explicitly set size in this way
  bool fcb_on = check_fcb_on(fcb);
  bool is_dir = check_is_dir(fcb);

  fcb->size = size;
  if (fcb_on) mark_fcb_on(fcb);
  else mark_fcb_off(fcb);

  if (is_dir) mark_is_dir(fcb);
  else mark_not_dir(fcb);
}

__device__ u32 get_size_of_fcb(FCB *fcb) {
  u32 mask = (1 << 31) + (1 << 30);
  return fcb->size & ~mask;
}

__device__ uchar * get_content(FileSystem *fs, int block_idx, int byte_offset) {
  // given a block index, get the pointer to the content of a file (or directory)
  return START_OF_CONTENTS + block_idx * fs->STORAGE_BLOCK_SIZE + byte_offset;
}

__device__ int get_fcb_by_name(FileSystem *fs, char *name, int dir_idx) {
  // get the **fcb index** by name **in the current directory**
  for (int i = 0; i < fs->FCB_ENTRIES; i++)
  {
    FCB *fcb = START_OF_FCB + i;
    if (check_fcb_on(fcb) && strmatch(fcb->filename, name) && fcb->dir_idx == dir_idx) {
      return i;
    }
  }
  return -1;
}

__device__ bool strmatch(char *start1, char* start2) {
  // match two strings, return true if they are the same
  int idx = 0;
  while (!(start1[idx] == '\0' || start2[idx] == '\0'))
  {
    if (start1[idx] != start2[idx])
    {
      return false;
    }
    idx++;
  }
  if (start1[idx] != start2[idx])
  {
    return false;
  }

  return true;
}


__device__ void my_memcpy(char *destination, char *source, int size) {
  // this will not automatically add '\0' to the string
  for (int i = 0; i < size; i++)
  {
    destination[i] = source[i];
  }
  
}

__device__ int my_strlen(char *s) {
  // find the length of a string, **including '\0'**
  int idx = 0;
  while (s[idx] != '\0')
  {
    idx++;
  }
  return idx+1;
}

__device__ bool file_exists_in_curr_dir(FileSystem *fs, char* s) {
  return get_fcb_by_name(fs, s, fs->cwd) != -1;
}

__device__ bool isdirname(char *s) {
  // check if a string is a directory name
  // a directory name ends with '/'
  int len = my_strlen(s);
  return s[len-2] == '/';
}

__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */
  // s ends with '\0'
  // op: open mode, G_READ or G_WRITE
  // returns the file pointer, which is the index of the FCB entry
  bool file_exists = file_exists_in_curr_dir(fs, s);
  switch (op)
  {
    case G_READ:
      // find file with the filename among all files, returns the index of the FCB
      if (file_exists) {
        return get_fcb_by_name(fs, s, fs->cwd);
      }
      assert(0);  // file not found
      break;
    case G_WRITE:
      // create the directory for the new file, if it's not already there; returns the address of the new FCB
      if (file_exists) {
        // have to empty the file in the next write operation
        // in which we will check the `size` attribute, if it's not 0, we will free the blocks
        int fcb_idx = get_fcb_by_name(fs, s, fs->cwd);
        // printf("fs_open file %s exists, index %d\n", s, fcb_idx);
        return fcb_idx;
      } else {  // file not exists
        gtime++;
        // allocate a new fcb index for the newly-created file
        for (int i = 0; i < fs->FCB_ENTRIES; i++)
        { // find an unused fcb
          FCB *target_fcb = START_OF_FCB + i;

          if (!check_fcb_on(target_fcb))
          {
            // mark the FCB as on and set its attrs
            target_fcb->dir_idx = fs->cwd;
            target_fcb->modified_time = gtime;
            set_size_of_fcb(target_fcb, 0);  // size at creation
            mark_fcb_on(target_fcb);
            mark_not_dir(target_fcb);
            target_fcb->creation_time = gtime;  // time at creation
            target_fcb->start_block_idx = 0;
            bool is_dir_name = isdirname(s);
            if (is_dir_name)
            {
              // this is a directory
              mark_is_dir(target_fcb);
              target_fcb->dir_idx = fs->cwd;
              s[my_strlen(s)-2] = '\0';  // remove the last '/'
              // printf("fs_open dir %s created, index %d\n", s, i);
              // printf("Parent dir idx %d\n", target_fcb->dir_idx);
            }

            // copy the filename
            int idx = 0;
            while (s[idx] != '\0')
            {
              target_fcb->filename[idx] = s[idx];
              idx++;
            }
            target_fcb->filename[idx] = '\0';

            if (is_dir_name && (target_fcb->dir_idx == -1))
            {
              // this is the root directory, no parent directory
              // printf("returning root dir fcb index %d\n", i);
              return i;
            }


            // add the filename to the directory file content
            uchar * cwd_content = get_content(fs, START_OF_FCB[fs->cwd].start_block_idx, 0);
            int cwd_curr_size = get_size_of_fcb(START_OF_FCB + fs->cwd);
            uchar * input = new uchar[cwd_curr_size + my_strlen(s)];

            my_memcpy((char*)input, (char*)cwd_content, cwd_curr_size);
            my_memcpy((char*)(input+cwd_curr_size), (char*)s, my_strlen(s));
            
            fs_write(fs, input, cwd_curr_size + my_strlen(s), fs->cwd);
            // printf("fs_open new fcb %s, index %d\n", s, i);
            
            delete[] input;
            return i;
          }
        }
        assert(0);  // no empty FCB
      }
      break;

    default:
      assert(0);  // no such option
      break;
  }
}

__device__ u32 block_of_bytes(FileSystem *fs, u32 bytes) {
  // returns how many blocks the `bytes` information will occupy
  u32 ret = bytes / fs->STORAGE_BLOCK_SIZE;
  if ((bytes % fs->STORAGE_BLOCK_SIZE) != 0) {
    ret++;
  }
  return ret;
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
  // fp the index of the FCB
  assert(check_fcb_on(START_OF_FCB + fp));
  uchar *start = START_OF_CONTENTS + START_OF_FCB[fp].start_block_idx * fs->STORAGE_BLOCK_SIZE;
  FCB fcb = START_OF_FCB[fp];   // the fcb for this file

  // printf("fs_read %d bytes from %s\n", size, fcb.filename);
  
  // read `size` bytes to buffer `output`
  for (u32 i = 0; i < size; i++)
  {
    output[i] = start[i];
  } 
}

__device__ void block_move(FileSystem *fs, int target_block_idx, int source_block_idx) {
  // printf("moving block %d to %d\n", source_block_idx, target_block_idx);

  uchar *target_start = START_OF_CONTENTS + target_block_idx * fs->STORAGE_BLOCK_SIZE;
  uchar *source_start = START_OF_CONTENTS + source_block_idx * fs->STORAGE_BLOCK_SIZE;
  for (int i = 0; i < fs->STORAGE_BLOCK_SIZE; i++)
  {
    target_start[i] = source_start[i];
    source_start[i] = 0;
  }
  mark_block_unused(fs, source_block_idx);
  mark_block_used(fs, target_block_idx);
}

__device__ u16 alloc_new_blocks(FileSystem *fs, int target_block_size) {
  // allocate contiguous blocks with `target_block_size`, register it in the bitmap
  // return the index of the first block
  // if no enough contiguous blocks, have to manage the fragmentation
  // printf("allocating %d blocks in alloc_new_blocks\n", target_block_size);
  int current_block_idx = 0;
  int block_count = 0;
  while (current_block_idx < STORAGE_BLOCK_COUNT)
  {
    if (check_block_used(fs, current_block_idx)) {
      // this block is used, reset the counter
      block_count = 0;
    } else {
      block_count++;
      if (block_count == target_block_size) {
        // found enough contiguous blocks
        // printf("contiguous block found, returning block %d, span %d\n", current_block_idx - target_block_size + 1, target_block_size);
        // mark blocks as used
        for (int i = 0; i < target_block_size; i++)
        {
          mark_block_used(fs, current_block_idx - target_block_size + 1 + i);
        }
        
        return current_block_idx - target_block_size + 1;
      }
    }
    current_block_idx++;
  }

  // printf("No enough contiguous blocks, have to manage the fragmentation\n");
  // not enough contiguous space, have to manage the fragmentation
  // compation algorithm
  int first_unused_block_idx = 0;
  while (true)
  {
    // find the first unused block idx
    while (first_unused_block_idx < STORAGE_BLOCK_COUNT)
    {
      if (!check_block_used(fs, first_unused_block_idx)) {
        break;
      }
      first_unused_block_idx++;
    }
    
    
    current_block_idx = first_unused_block_idx+1;
    // find the next used block idx
    while (current_block_idx < STORAGE_BLOCK_COUNT)
    {
      if (check_block_used(fs, current_block_idx)) {
        break;
      }
      current_block_idx++;
    }

    if (current_block_idx >= STORAGE_BLOCK_COUNT) {
      // no more used blocks
      break;
    }

    // swap the two blocks
    block_move(fs, first_unused_block_idx, current_block_idx);
    // reassign the associated fcbs of the moved block
    for (int i = 0; i < fs->FCB_ENTRIES; i++)
    {
      FCB *fcb = START_OF_FCB + i;
      if (fcb->start_block_idx == current_block_idx)
      {
        // printf("reassigning fcb block %d to %d\n", fcb->start_block_idx, first_unused_block_idx);
        fcb->start_block_idx = first_unused_block_idx;
        break;
      }
    }
    
    first_unused_block_idx++;
    current_block_idx++;
    if (current_block_idx >= STORAGE_BLOCK_COUNT) {
      // no more used blocks
      break;
    }
  }

  // reallocate
  // printf("reallocating %d blocks in alloc_new_blocks\n", target_block_size);
  current_block_idx = 0;
  block_count = 0;
  while (current_block_idx < STORAGE_BLOCK_COUNT)
  {
    if (check_block_used(fs, current_block_idx)) {
      // this block is used, reset the counter
      block_count = 0;
    } else {
      block_count++;
      if (block_count == target_block_size) {
        // found enough contiguous blocks
        // printf("contiguous block found, returning block %d, span %d\n", current_block_idx - target_block_size + 1, target_block_size);
        // mark blocks as used
        for (int i = 0; i < target_block_size; i++)
        {
          mark_block_used(fs, current_block_idx - target_block_size + 1 + i);
        }
        
        return current_block_idx - target_block_size + 1;
      }
    }
    current_block_idx++;
  }
  assert(0);  // fail to reallocate

}



__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	// write bytes to the file
  // fp the index of the FCB
  gtime++;

  uchar *start = START_OF_CONTENTS + START_OF_FCB[fp].start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
  FCB *fcb = START_OF_FCB+fp;   // the fcb for this file
  u16 start_block_idx = fcb->start_block_idx;
  
  fcb->modified_time = gtime;
  // printf("fs_write %d bytes into %s\n", size, fcb->filename);
  // printf("start_block_idx %d\n", start_block_idx);
  // printf("check used: %d\n", check_block_used(fs, start_block_idx));

  // if the file already exists, we have to free the blocks 
  for (u32 i = 0; i < block_of_bytes(fs, get_size_of_fcb(fcb)); i++)
  {
    mark_block_unused(fs, start_block_idx+i);
  }
  // empty the bytes, replace by 0
  for (u32 i = 0; i < get_size_of_fcb(fcb); i++)
  {
    start[i] = 0;
  }


  // begin writing to new file
  bool can_directly_write = true;
  for (u32 i = 0; i < block_of_bytes(fs, size); i++)
  {
    if (check_block_used(fs, start_block_idx+i))
    {
      can_directly_write = false;
      break;
    }
  }
  
  if (can_directly_write)
  {
    // printf("directly writing %d blocks starting from block %d\n", block_of_bytes(fs, size), start_block_idx);
    // directly write to it
    for (u32 i = 0; i < size; i++)
    {
      start[i] = input[i];
    }
    for (u32 i = 0; i < block_of_bytes(fs, size); i++)
    {
      mark_block_used(fs, start_block_idx+i);
    }
    set_size_of_fcb(fcb, size);
    
    return size;
    
  } else {
    // cannot directly write, need to fix fragmentation, then directly write
    fcb->start_block_idx = alloc_new_blocks(fs, block_of_bytes(fs, size));
    // printf("resetting start_block_idx to %d\n", fcb->start_block_idx);
    // perform write
    start = START_OF_CONTENTS + fcb->start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
    for (u32 i = 0; i < size; i++)
    {
      start[i] = input[i];
    }
    for (u32 i = 0; i < block_of_bytes(fs, size); i++)
    {
      mark_block_used(fs, fcb->start_block_idx+i);
    }

    set_size_of_fcb(fcb, size);
    return size;
  }
}

__device__ void pwd_helper(FileSystem *fs, int fcb_idx) {
  // to print cwd, this should be called `pwd_helper(fs, fs->cwd)`

  // recursively print the path of the current directory
  FCB *fcb = START_OF_FCB + fcb_idx;
  if (fcb->dir_idx == -1) {
    // root directory
    return;
  }
  pwd_helper(fs, fcb->dir_idx);
  printf("/%s", fcb->filename);
}


// ls_d, ls_s, cd_p, pwd goes here
__device__ void fs_gsys(FileSystem *fs, int op)
{
  FCB *cwd_fcb = START_OF_FCB + fs->cwd;
  uchar *cwd_content = get_content(fs, cwd_fcb->start_block_idx, 0);
  // get the number of files and subdirectories **in the current directory**
  int file_count = 0;
  for (int i = 0; i < get_size_of_fcb(cwd_fcb); i++)
  {
    if (cwd_content[i] == '\0')
    {
      file_count++;
    }
  }
  
  // printf("number of files or dirs: %d\n", file_count);

	/* Implement ls_d, ls_s, cd_p, pwd operation here */
  switch (op)
  {
  case PWD:
  {
    // printf("printing pwd\n");
    // printf("fs.cwd: %d\n", fs->cwd);
    if (cwd_fcb->dir_idx == -1) {
      // root directory
      printf("/\n");
      break;
    } else {
      pwd_helper(fs, fs->cwd);
      printf("\n");
      break;
    }
  }
  case CD_P:
  {
    // cd to parent dir
    if (cwd_fcb->dir_idx == -1) {
      // root directory
      // printf("already in root directory\n");
      break;
    } else {
      // printf("changed directory to parent directory\n");
      fs->cwd = cwd_fcb->dir_idx;
      break;
    }
  }
  case LS_D:
  {

    // list the file name and order by modified time of files

    printf("===sort by modified time===\n");

    int last_item_time = (1<<15); // trace the time of last printed file
    // print the most recent modified file before the last item
    for (int i = 0; i < file_count; i++)  // print each file once
    {
      int latest_modified_time = 0;
      FCB latest_fcb;

      // tokenize the file name in the cwd content
      uchar current_byte;
      int token_start_idx = 0;

      // find the most recently modified file or subdir
      for (int j = 0; j < get_size_of_fcb(cwd_fcb); j++)
      {
        current_byte = *get_content(fs, cwd_fcb->start_block_idx, j);
        if (current_byte == '\0')
        {
          // get this full token
          char *token = new char[20];
          my_memcpy(token, (char*)get_content(fs, cwd_fcb->start_block_idx, token_start_idx), j-token_start_idx+1);
          token[j-token_start_idx+1] = '\0';
          // printf("token: %s\n", token);
          // get the fcb
          int fcb_idx = get_fcb_by_name(fs, token, fs->cwd);
          FCB *fcb = START_OF_FCB + fcb_idx;
          // printf("examining fcb %s, size %d\n", fcb->filename, fcb->size);
          if (check_fcb_on(fcb) && (fcb->modified_time > latest_modified_time) && (fcb->modified_time < last_item_time))
          {
            latest_fcb = *fcb;
            latest_modified_time = fcb->modified_time;
          }
          
          token_start_idx = j+1;
          delete[] token;
        }
        
      }

      last_item_time = latest_fcb.modified_time;
      if (check_is_dir(&latest_fcb))
      {
        printf("%s d\n", latest_fcb.filename);
      } else {
        printf("%s\n", latest_fcb.filename);
      }
      
      // printf("%s   time%d\n", latest_fcb.filename, last_item_time);
    }
    break;
  }

  case LS_S:
  {
      
    printf("===sort by file size===\n");
    // If there are several files with the same size, then first create first print.

    
    u32 last_item_size = (1<<31); // the distinct size of the last printed file
    int print_count = 0;

    while (print_count < file_count)
    {

      // tokenize the file name in the cwd content
      uchar current_byte;
      int token_start_idx = 0;

      int largest_file_size = 0;
      // printf("last item size is: %d\n", last_item_size);
      // get the largest file size less than `last_item_size`
      for (int j = 0; j < get_size_of_fcb(cwd_fcb); j++)
      {
        current_byte = *get_content(fs, cwd_fcb->start_block_idx, j);
        if (current_byte == '\0')
        {
          // get this full token
          char *token = new char[20];
          my_memcpy(token, (char*)get_content(fs, cwd_fcb->start_block_idx, token_start_idx), j-token_start_idx+1);
          token[j-token_start_idx+1] = '\0';
          // get the fcb
          int fcb_idx = get_fcb_by_name(fs, token, fs->cwd);
          FCB *fcb = START_OF_FCB + fcb_idx;
          // printf("examining token %s, size is %d\n", fcb->filename, get_size_of_fcb(fcb));
          if (check_fcb_on(fcb) && (get_size_of_fcb(fcb) > largest_file_size) && (get_size_of_fcb(fcb) < last_item_size))
          {
            largest_file_size = get_size_of_fcb(fcb);
          }
          
          token_start_idx = j+1;
          delete[] token;
        }
        
      }
      last_item_size = largest_file_size;

      // printf("largest file size: %d\n", largest_file_size);

      // count the number of files with the size of largest_file_size
      token_start_idx = 0;
      int largest_file_count = 0;
      for (int j = 0; j < get_size_of_fcb(cwd_fcb); j++)
      {
        current_byte = *get_content(fs, cwd_fcb->start_block_idx, j);
        if (current_byte == '\0')
        {
          // get this full token
          char *token = new char[20];
          my_memcpy(token, (char*)get_content(fs, cwd_fcb->start_block_idx, token_start_idx), j-token_start_idx+1);
          token[j-token_start_idx+1] = '\0';
          // get the fcb
          int fcb_idx = get_fcb_by_name(fs, token, fs->cwd);
          FCB *fcb = START_OF_FCB + fcb_idx;
          // printf("examining token %s, size is %d\n", fcb->filename, get_size_of_fcb(fcb));
          if (check_fcb_on(fcb) && (get_size_of_fcb(fcb) == largest_file_size))
          {
            largest_file_count++;
          }
          token_start_idx = j+1;
          delete[] token;
        }
        
      }
      // printf("largest file size: %d, count: %d\n", largest_file_size, largest_file_count);

      // now we have the size, find the file or subdir with the same size and print by creation time order
      u16 last_item_time = 0;
      for (int i = 0; i < largest_file_count; i++)
      {
        // find the file with the file size of largest_file_size and the earliest created time among all unprinted items

        u16 earliest_created_time = (1<<15);
        FCB *earliest_fcb;
        token_start_idx = 0;
        for (int j = 0; j < get_size_of_fcb(cwd_fcb); j++)
        {
          current_byte = *get_content(fs, cwd_fcb->start_block_idx, j);
          // printf("examining byte %c\n", current_byte);
          if (current_byte == '\0')
          {
            // get this full token
            char *token = new char[20];
            my_memcpy(token, (char*)get_content(fs, cwd_fcb->start_block_idx, token_start_idx), j-token_start_idx+1);
            token[j-token_start_idx+1] = '\0';

            // printf("examining token %s\n", token);
            // get the fcb
            int fcb_idx = get_fcb_by_name(fs, token, fs->cwd);
            FCB *fcb = START_OF_FCB + fcb_idx;
            // printf("last item time: %d\n", last_item_time);
            if (check_fcb_on(fcb) && (get_size_of_fcb(fcb) == largest_file_size) && (fcb->creation_time < earliest_created_time) && (fcb->creation_time > last_item_time))
            {
              earliest_fcb = fcb;
              earliest_created_time = fcb->creation_time;
            }
            
            token_start_idx = j+1;
            delete[] token;
          }
        }
        last_item_time = earliest_fcb->creation_time;
        if (check_is_dir(earliest_fcb))
        {
          printf("%s %d d\n", earliest_fcb->filename, get_size_of_fcb(earliest_fcb));
        } else {
          printf("%s %d\n", earliest_fcb->filename, get_size_of_fcb(earliest_fcb));
        }
      }
      print_count += largest_file_count;
    }
    break;
  }
  default:
    assert(0);
    break;  // no such option
  } // end of switch
  
}

__device__ void remove_file(FileSystem *fs, FCB *target_fcb) {
      mark_fcb_off(target_fcb);

      // remove the item in the parent dir's content
      {
        // printf("removing %s in dir %s\n", target_fcb->filename, START_OF_FCB[fs->cwd].filename);

        uchar * cwd_content = get_content(fs, START_OF_FCB[fs->cwd].start_block_idx, 0);
        int cwd_curr_size = get_size_of_fcb(START_OF_FCB + fs->cwd);
        
        uchar *new_input = new uchar[cwd_curr_size];
        my_memcpy((char*)new_input, (char*)cwd_content, cwd_curr_size);

        // find the position of this filename
        uchar current_byte;
        int token_start_idx = 0;
        for (int j = 0; j < cwd_curr_size; j++)
        {
          current_byte = new_input[j];
          // printf("rm I am examining byte %c\n", current_byte);
          if (current_byte == '\0')
          {
            // printf("token starts with %c\n", *(new_input+token_start_idx));
            // printf("token is %s\n", new_input+token_start_idx);
            if (strmatch(target_fcb->filename, (char*)(new_input+token_start_idx)))
            {
              // printf("match\n");
              my_memcpy((char*)(new_input+token_start_idx), (char*)(new_input+j+1), cwd_curr_size-j-1);
              break;
            }
            token_start_idx = j+1;
          }
        }

        fs_write(fs, new_input, cwd_curr_size-my_strlen(target_fcb->filename), fs->cwd);
        delete[] new_input;
      }

      // free the content memory
      uchar *start = get_content(fs, target_fcb->start_block_idx, 0); // the initial byte of the file content
      // printf("fs_delete removing %d bytes of %s, start from block %d span %d\n", get_size_of_fcb(target_fcb), target_fcb->filename, target_fcb->start_block_idx, block_of_bytes(fs, get_size_of_fcb(target_fcb)));

      // free the blocks  
      for (u32 i = 0; i < block_of_bytes(fs, get_size_of_fcb(target_fcb)); i++)
      {
        mark_block_unused(fs, target_fcb->start_block_idx+i);
      }
      // empty the bytes, replace by 0
      for (u32 i = 0; i < get_size_of_fcb(target_fcb); i++)
      {
        start[i] = 0;
      }
    
}

// rm, cd, mkdir, rm_rf goes here
__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
  // absolute path operation support
  if (s[0] == '/')
  {
    // first cd to root
    int token_start_idx = 1;
    int curr_cwd = fs->cwd;
    fs->cwd = 0;
    if (my_strlen(s) == 2)
    {
      return;
    }
    
    char *token = new char[20];
    // cd to the last dir
    for (int x = 1; x < my_strlen(s); x++)
    {
      if (s[x] == '/')
      {
        my_memcpy(token, s+token_start_idx, x-token_start_idx);
        token[x-token_start_idx] = '\0';
        // printf("cd to %s\n", token);
        fs_gsys(fs, CD, token);
        token_start_idx = x+1;
      }
      
      if (s[x] == '\0')
      {
        my_memcpy(token, s+token_start_idx, x-token_start_idx);
        token[x-token_start_idx] = '\0';
        token_start_idx = x+1;
        break;
      }
    }

    fs_gsys(fs, op, token);
    delete[] token;

    // cd back
    if (op != CD)
    {
      fs->cwd = curr_cwd;
    }
    return;
  }

  // find the specific file in the FCB
  bool file_exists = file_exists_in_curr_dir(fs, s);
  int fcb_idx = 0;
  if (file_exists)
  {
    fcb_idx = get_fcb_by_name(fs, s, fs->cwd);
  }
  FCB *target_fcb = START_OF_FCB + fcb_idx;
  

	/* Implement rm operation here */
  if (op == RM)
  {
    // delete the specific file
    if (!file_exists)
    {
      assert(0);  // file not found
    } else {
      remove_file(fs, target_fcb);
    }
    
  } else if (op == MKDIR)
  {
    // create a new directory
    if (file_exists) {
      return; // allow existing directory
    } else {
      // append '/' at the end of s
      int len = my_strlen(s);
      char *tmp = new char[21];
      my_memcpy(tmp, s, len);
      tmp[len-1] = '/';
      tmp[len] = '\0';
      fs_open(fs, tmp, G_WRITE);
      delete[] tmp;
    }
  } else if (op == CD) {
    assert(file_exists);  // if assertion failed, the directory does not exist
    fs->cwd = fcb_idx;
    // printf("change directory to %s, index %d\n", s, fs->cwd);
  } else if (op == RM_RF) {
    // Remove the app directory and all its subdirectories and files recursively
    assert(file_exists);  // if assertion failed, the directory or file does not exist
    // printf("RM_RF called, to remove %s\n", s);

    // locate the additional fcb
    if (!check_is_dir(target_fcb))
    {
      // remove that regular file only
      // printf("rm_rf removing regular file %s\n", s);
      remove_file(fs, target_fcb);
    } else {
      // recursively remove the directory, its subdirectories and files

      // first CD into the target dir, after it has been removed CD back
      fs->cwd = target_fcb - START_OF_FCB;

      uchar current_byte;
      int current_idx = 0;

      // token start idx will always be 0
      while (get_content(fs, START_OF_FCB[fs->cwd].start_block_idx, 0)[0] != '\0')
      {
        current_byte = *get_content(fs, target_fcb->start_block_idx, current_idx);
        // printf("I am examining byte %c\n", current_byte);
        if (current_byte == '\0')
        {
          // get this full token
          char *token = new char[20];
          my_memcpy(token, (char*)get_content(fs, target_fcb->start_block_idx, 0), current_idx+1);
          // printf("token is %s\n", token);

          FCB *token_fcb = START_OF_FCB + get_fcb_by_name(fs, token, fs->cwd);
          if (check_is_dir(token_fcb))
          {
            // recursively remove the subdirectory
            fs_gsys(fs, RM_RF, token);
          } else {
            // remove the regular file
            remove_file(fs, token_fcb);
          }
          
          current_idx = -1;
          delete[] token;
        }
        current_idx++;
      }
      
      // CD back
      // cd to parent dir
      fs->cwd = target_fcb->dir_idx;

      // since we have emptied the dir's subunits, we can now remove it as a regular file
      remove_file(fs, target_fcb);

    }
    
  }
  
  
}
