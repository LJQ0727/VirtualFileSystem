#include "hip/hip_runtime.h"
﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

__device__ __managed__ u32 gtime = 0;

__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS,
              FCB *start_of_fcb)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
  fs->STORAGE_BLOCK_COUNT = (fs->STORAGE_SIZE - fs->SUPERBLOCK_SIZE - (fs->FCB_SIZE * fs->FCB_ENTRIES)) / fs->STORAGE_BLOCK_SIZE;

  // init some custom pointers
  fs->start_of_fcb = start_of_fcb;
  fs->start_of_superblock = volume;
  fs->start_of_contents = volume + FILE_BASE_ADDRESS;
  fs->cwd = -1;   // the root directory will have parent dir index -1

  // initialize volume
  for (int i = 0; i < VOLUME_SIZE; i++)
    volume[i] = 0;

  // make root directory and cd to it
  fs_gsys(fs, MKDIR, "/\0");
  fs_gsys(fs, CD, "/\0");
    
}
__device__ bool strmatch(char *start1, char* start2);

__device__ void mark_fcb_on(FCB *fcb)
{
  fcb->size |= (1 << 31);
}

__device__ void mark_fcb_off(FCB *fcb)
{
  fcb->size &= ~(1 << 31);
}

__device__ bool check_fcb_on(FCB *fcb)
{
  // the fcb is on if its MSB of size is 1
  return fcb->size & (1<<31);
}

__device__ void mark_is_dir(FCB *fcb) {
  fcb->size |= (1 << 30);
}

__device__ void mark_not_dir(FCB *fcb) {
  fcb->size &= ~(1 << 30);
}

__device__ bool check_is_dir(FCB *fcb) {
  return fcb->size & (1<<31);
}


__device__ uchar * get_content(FileSystem *fs, int block_idx, int byte_offset) {
  // given a block index, get the pointer to the content of a file (or directory)
  return fs->start_of_contents + block_idx * fs->STORAGE_BLOCK_SIZE + byte_offset;
}

__device__ int get_fcb_by_name(FileSystem *fs, char *name) {
  // get the fcb index by name
  for (int i = 0; i < fs->FCB_ENTRIES; i++)
  {
    FCB *fcb = fs->start_of_fcb + i;
    if (check_fcb_on(fcb) && strmatch(fcb->filename, name)) {
      return i;
    }
  }
  assert(0);  // no such named fcb
  return -1;
}

__device__ void set_gtime_recursive(FileSystem *fs, int fcb_idx, u32 gtime) {
  // recursively set the gtime of the fcb and propagate this change to all parent dirs
  FCB *fcb = fs->start_of_fcb + fcb_idx;
  fcb->modified_time = gtime;
  if (fcb->parent_dir_idx != -1) {
    set_gtime_recursive(fs, fcb->dir_idx, gtime);
    printf("modified gtime of %s: %d isdir: %d\n", fcb->filename ,gtime, fcb->is_dir);
  }
}
  

__device__ bool strmatch(char *start1, char* start2) {
  // match two strings, return true if they are the same
  int idx = 0;
  while (!(start1[idx] == '\0' || start2[idx] == '\0'))
  {
    if (start1[idx] != start2[idx])
    {
      return false;
    }
    idx++;
  }
  if (start1[idx] != start2[idx])
  {
    return false;
  }

  return true;
}

__device__ void mark_block_used(FileSystem *fs, int block_idx) {
  // mark a block as used in the superblock
  // operate on only one block at a time
  uchar bitmap = fs->start_of_superblock[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  fs->start_of_superblock[block_idx/8] = bitmap | mask;
}

__device__ void mark_block_unused(FileSystem *fs, int block_idx) {
  // mark a block as unused in the superblock
  // operate on only one block at a time
  uchar bitmap = fs->start_of_superblock[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  fs->start_of_superblock[block_idx/8] = bitmap & ~mask;
}

__device__ bool check_block_used(FileSystem *fs, int block_idx) {
  uchar bitmap = fs->start_of_superblock[block_idx/8];
  uchar mask = 1 << (block_idx % 8);
  return bitmap & mask;
}


__device__ void my_memcpy(char *destination, char *source, int size) {
  // this will not automatically add '\0' to the string
  for (int i = 0; i < size; i++)
  {
    destination[i] = source[i];
  }
  
}

__device__ int my_strlen(char *s) {
  // find the length of a string, **including '\0'**
  int idx = 0;
  while (s[idx] != '\0')
  {
    idx++;
  }
  return idx+1;
}

__device__ bool file_exists_in_curr_dir(FileSystem *fs, char* s) {
  bool file_exists = false;
  // find if the specific file already exists in the FCB
  if (s[0] == '/')
  {
    return; // abs path not implemented
  } else {
    // find the file in the current directory
    for (int i = 0; i < fs->start_of_fcb[fs->cwd].size - my_strlen(s); i++)
    {
      if (strmatch((char*)get_content(fs, fs->start_of_fcb[fs->cwd].start_block_idx, i), s))
      {
        if (i == 0 || (*get_content(fs, fs->start_of_fcb[fs->cwd].start_block_idx, i-1) == '\0'))
        {
          // found the file
          file_exists = true;
          break;
        }
      }
    }
  }
  return file_exists;
  printf("file_exists for %s in current dir is %d\n", s, file_exists);

}

__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */
  // s ends with '\0'
  // op: open mode, G_READ or G_WRITE
  // returns the file pointer, which is the index of the FCB entry
  gtime++;
  bool file_exists = file_exists_in_curr_dir(fs, s);

  
  switch (op)
  {
    case G_READ:
      // find file with the filename among all files, returns the index of the FCB
      if (file_exists) {
        return get_fcb_by_name(fs, s);
      }
      assert(0);  // file not found
      break;
    case G_WRITE:
      // create the directory for the new file, if it's not already there; returns the address of the new FCB
      if (file_exists) {
        // have to empty the file in the next write operation
        // in which we will check the `size` attribute, if it's not 0, we will free the blocks
        int fcb_idx = get_fcb_by_name(fs, s);
        set_gtime_recursive(fs, fcb_idx, gtime);
        printf("fs_open file %s exists, index %d\n", s, fcb_idx);
        return fcb_idx;
      } else {  // file not exists
        // allocate a new fcb index for the newly-created file
        for (int i = 0; i < fs->FCB_ENTRIES; i++)
        { // find an unused fcb
          FCB target_fcb = fs->start_of_fcb[i];
          if (!target_fcb.is_on)
          {
            // mark the FCB as on and set its attrs
            fs->start_of_fcb[i].is_on = true;
            fs->start_of_fcb[i].modified_time = gtime;
            fs->start_of_fcb[i].size = 0;  // size at creation
            fs->start_of_fcb[i].creation_time = gtime;  // time at creation
            fs->start_of_fcb[i].start_block_idx = 0;
            fs->start_of_fcb[i].is_dir = false;
            fs->start_of_fcb[i].parent_dir_idx = fs->start_of_fcb[fs->cwd].parent_dir_idx;
            fs->start_of_fcb[i].dir_idx = fs->cwd;

            // copy the filename
            int idx = 0;
            while (s[idx] != '\0')
            {
              fs->start_of_fcb[i].filename[idx] = s[idx];
              idx++;
            }
            fs->start_of_fcb[i].filename[idx] = '\0';


            // add the filename to the directory file content
            uchar * cwd_content = get_content(fs, fs->start_of_fcb[fs->cwd].start_block_idx, 0);
            int cwd_curr_size = fs->start_of_fcb[fs->cwd].size;
            uchar * input = new uchar[cwd_curr_size + my_strlen(s)];

            my_memcpy((char*)input, (char*)cwd_content, cwd_curr_size);
            my_memcpy((char*)(input+cwd_curr_size), (char*)s, my_strlen(s));
            
            fs_write(fs, input, cwd_curr_size + my_strlen(s), fs->cwd);
            
            printf("fs_open new fcb %s, index %d\n", s, i);
            set_gtime_recursive(fs, i, gtime);
            return i;
          }
        }
        assert(0);  // no empty FCB
      }
      break;

    default:
      assert(0);  // no such option
      break;
  }
}

__device__ u32 block_of_bytes(FileSystem *fs, u32 bytes) {
  // returns how many blocks the `bytes` information will occupy
  u32 ret = bytes / fs->STORAGE_BLOCK_SIZE;
  if ((bytes % fs->STORAGE_BLOCK_SIZE) != 0) {
    ret++;
  }
  return ret;
}

__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
  // fp the index of the FCB
  assert(fs->start_of_fcb[fp].is_on);
  uchar *start = fs->start_of_contents + fs->start_of_fcb[fp].start_block_idx * fs->STORAGE_BLOCK_SIZE;
  FCB fcb = fs->start_of_fcb[fp];   // the fcb for this file

  printf("fs_read %d bytes from %s\n", size, fcb.filename);
  
  // read `size` bytes to buffer `output`
  for (u32 i = 0; i < size; i++)
  {
    output[i] = start[i];
  } 
}

__device__ void block_move(FileSystem *fs, int target_block_idx, int source_block_idx) {
  printf("moving block %d to %d\n", source_block_idx, target_block_idx);

  uchar *target_start = fs->start_of_contents + target_block_idx * fs->STORAGE_BLOCK_SIZE;
  uchar *source_start = fs->start_of_contents + source_block_idx * fs->STORAGE_BLOCK_SIZE;
  for (int i = 0; i < fs->STORAGE_BLOCK_SIZE; i++)
  {
    target_start[i] = source_start[i];
    source_start[i] = 0;
  }
  mark_block_unused(fs, source_block_idx);
  mark_block_used(fs, target_block_idx);
}

__device__ u16 alloc_new_blocks(FileSystem *fs, int target_block_size) {
  // allocate contiguous blocks with `target_block_size`, register it in the bitmap
  // return the index of the first block
  // if no enough contiguous blocks, have to manage the fragmentation
  printf("allocating %d blocks in alloc_new_blocks\n", target_block_size);
  int current_block_idx = 0;
  int block_count = 0;
  while (current_block_idx < fs->STORAGE_BLOCK_COUNT)
  {
    if (check_block_used(fs, current_block_idx)) {
      // this block is used, reset the counter
      block_count = 0;
    } else {
      block_count++;
      if (block_count == target_block_size) {
        // found enough contiguous blocks
        printf("contiguous block found, returning block %d, span%d\n", current_block_idx - target_block_size + 1, target_block_size);
        // mark blocks as used
        for (int i = 0; i < target_block_size; i++)
        {
          mark_block_used(fs, current_block_idx - target_block_size + 1 + i);
        }
        
        return current_block_idx - target_block_size + 1;
      }
    }
    current_block_idx++;
  }

  printf("No enough contiguous blocks, have to manage the fragmentation\n");
  // not enough contiguous space, have to manage the fragmentation
  // compation algorithm
  int first_unused_block_idx = 0;
  while (true)
  {
    // find the first unused block idx
    while (first_unused_block_idx < fs->STORAGE_BLOCK_COUNT)
    {
      if (!check_block_used(fs, first_unused_block_idx)) {
        break;
      }
      first_unused_block_idx++;
    }
    
    
    current_block_idx = first_unused_block_idx+1;
    // find the next used block idx
    while (current_block_idx < fs->STORAGE_BLOCK_COUNT)
    {
      if (check_block_used(fs, current_block_idx)) {
        break;
      }
      current_block_idx++;
    }

    if (current_block_idx >= fs->STORAGE_BLOCK_COUNT) {
      // no more used blocks
      break;
    }

    // swap the two blocks
    block_move(fs, first_unused_block_idx, current_block_idx);
    // reassign the associated fcbs of the moved block
    for (int i = 0; i < fs->FCB_ENTRIES; i++)
    {
      FCB *fcb = fs->start_of_fcb + i;
      if (fcb->start_block_idx == current_block_idx)
      {
        printf("reassigning fcb block %d to %d\n", fcb->start_block_idx, first_unused_block_idx);
        fcb->start_block_idx = first_unused_block_idx;
        break;
      }
    }
    
    first_unused_block_idx++;
    current_block_idx++;
    if (current_block_idx >= fs->STORAGE_BLOCK_COUNT) {
      // no more used blocks
      break;
    }
  }

  // reallocate
  printf("reallocating %d blocks in alloc_new_blocks\n", target_block_size);
  current_block_idx = 0;
  block_count = 0;
  while (current_block_idx < fs->STORAGE_BLOCK_COUNT)
  {
    if (check_block_used(fs, current_block_idx)) {
      // this block is used, reset the counter
      block_count = 0;
    } else {
      block_count++;
      if (block_count == target_block_size) {
        // found enough contiguous blocks
        printf("contiguous block found, returning block %d, span%d\n", current_block_idx - target_block_size + 1, target_block_size);
        // mark blocks as used
        for (int i = 0; i < target_block_size; i++)
        {
          mark_block_used(fs, current_block_idx - target_block_size + 1 + i);
        }
        
        return current_block_idx - target_block_size + 1;
      }
    }
    current_block_idx++;
  }
  assert(0);  // fail to reallocate

}



__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	// write bytes to the file
  // fp the index of the FCB
  gtime++;
  set_gtime_recursive(fs, fp, gtime);

  uchar *start = fs->start_of_contents + fs->start_of_fcb[fp].start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
  FCB *fcb = fs->start_of_fcb+fp;   // the fcb for this file
  u16 start_block_idx = fcb->start_block_idx;
  
  printf("fs_write %d bytes into %s\n", size, fcb->filename);
  // printf("start_block_idx %d\n", start_block_idx);
  // printf("fcb->size %d\n", fcb->size);
  // printf("check used: %d\n", check_block_used(fs, start_block_idx));

  // if the file already exists, we have to free the blocks 
  for (u32 i = 0; i < block_of_bytes(fs, fcb->size); i++)
  {
    mark_block_unused(fs, start_block_idx+i);
  }
  // empty the bytes, replace by 0
  for (u32 i = 0; i < fcb->size; i++)
  {
    start[i] = 0;
  }


  // begin writing to new file
  bool can_directly_write = true;
  for (u32 i = 0; i < block_of_bytes(fs, size); i++)
  {
    if (check_block_used(fs, start_block_idx+i))
    {
      can_directly_write = false;
      break;
    }
  }
  
  if (can_directly_write)
  {
    printf("directly writing %d blocks starting from block %d\n", block_of_bytes(fs, size), start_block_idx);
    // directly write to it
    for (u32 i = 0; i < size; i++)
    {
      start[i] = input[i];
    }
    for (u32 i = 0; i < block_of_bytes(fs, size); i++)
    {
      mark_block_used(fs, start_block_idx+i);
    }
    fcb->size = size;
    
    return size;
    
  } else {
    // cannot directly write, need to fix fragmentation, then directly write
    fcb->start_block_idx = alloc_new_blocks(fs, block_of_bytes(fs, size));
    printf("cannot directly write, resetting start_block_idx to %d\n", fcb->start_block_idx);
    // perform write
    start = fs->start_of_contents + fcb->start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
    for (u32 i = 0; i < size; i++)
    {
      start[i] = input[i];
    }
    for (u32 i = 0; i < block_of_bytes(fs, size); i++)
    {
      mark_block_used(fs, fcb->start_block_idx+i);
    }

    fcb->size = size;
    return size;
  }
}

__device__ void pwd_helper(FileSystem *fs, int fcb_idx) {
  // to print cwd, this should be called `pwd_helper(fs, fs->cwd)`

  // recursively print the path of the current directory
  FCB *fcb = fs->start_of_fcb + fcb_idx;
  if (fcb->parent_dir_idx == -1) {
    // root directory
    return;
  }
  pwd_helper(fs, fcb->parent_dir_idx);
  printf("/%s", fcb->filename);
}


// ls_d, ls_s, cd_p, pwd goes here
__device__ void fs_gsys(FileSystem *fs, int op)
{
  FCB cwd_fcb = fs->start_of_fcb[fs->cwd];
  uchar *cwd_content = get_content(fs, cwd_fcb.start_block_idx, 0);
  // get the number of files and subdirectories **in the current directory**
  int file_count = 0;
  for (int i = 0; i < cwd_fcb.size; i++)
  {
    if (cwd_content[i] == '\0')
    {
      file_count++;
    }
  }
  
  // printf("number of files or dirs: %d\n", file_count);

	/* Implement ls_d, ls_s, cd_p, pwd operation here */
  switch (op)
  {
  case PWD:
  {
    printf("printing pwd\n");
    // printf("fs.cwd: %d\n", fs->cwd);
    if (cwd_fcb.parent_dir_idx == -1) {
      // root directory
      printf("/\n");
      break;
    } else {
      pwd_helper(fs, fs->cwd);
      printf("\n");
      break;
    }
  }
  case LS_D:
  {

    // list the file name and order by modified time of files

    printf("===sort by modified time===\n");

    int last_item_time = (1<<15); // trace the time of last printed file
    // print the most recent modified file before the last item
    for (int i = 0; i < file_count; i++)  // print each file once
    {
      int latest_modified_time = 0;
      FCB latest_fcb;
      int latest_fcb_idx = -1;

      // tokenize the file name in the cwd content
      uchar current_byte;
      int token_start_idx = 0;

      // find the most recently modified file or subdir
      for (int j = 0; j < cwd_fcb.size; j++)
      {
        current_byte = *get_content(fs, cwd_fcb.start_block_idx, j);
        if (current_byte == '\0')
        {
          // get this full token
          char token[21];
          my_memcpy(token, (char*)get_content(fs, cwd_fcb.start_block_idx, token_start_idx), 21);
          // get the fcb
          int fcb_idx = get_fcb_by_name(fs, token);
          FCB *fcb = fs->start_of_fcb + fcb_idx;
          if (check_fcb_on(fcb) && (fcb->modified_time > latest_modified_time) && (fcb->modified_time < last_item_time))
          {
            latest_fcb = *fcb;
            latest_modified_time = fcb->modified_time;
            latest_fcb_idx = fcb_idx;
          }
          
          token_start_idx = j+1;
        }
        
      }

      last_item_time = latest_fcb.modified_time;
      if (latest_fcb.is_dir)
      {
        printf("%s d\n", latest_fcb.filename);
      } else {
        printf("%s\n", latest_fcb.filename);
      }
      
      // printf("%s   time%d\n", latest_fcb.filename, last_item_time);
    }
    break;
  }

  case LS_S:
  {
      
    printf("===sort by file size===\n");
    // If there are several files with the same size, then first create first print.

    
    u16 last_item_size = (1<<15); // the distinct size of the last printed file
    int print_count = 0;

    while (print_count < file_count)
    {

      // tokenize the file name in the cwd content
      uchar current_byte;
      int token_start_idx = 0;

      int largest_file_size = 0;
      // get the largest file size less than `last_item_size`
      for (int j = 0; j < cwd_fcb.size; j++)
      {
        current_byte = *get_content(fs, cwd_fcb.start_block_idx, j);
        if (current_byte == '\0')
        {
          // get this full token
          char token[21];
          my_memcpy(token, (char*)get_content(fs, cwd_fcb.start_block_idx, token_start_idx), 21);
          // get the fcb
          int fcb_idx = get_fcb_by_name(fs, token);
          FCB *fcb = fs->start_of_fcb + fcb_idx;
          if (check_fcb_on(fcb) && (fcb->size > largest_file_size) && (fcb->modified_time < last_item_size))
          {
            largest_file_size = fcb->size;
          }
          
          token_start_idx = j+1;
        }
        
      }
      last_item_size = largest_file_size;

      // printf("largest file size: %d\n", largest_file_size);

      // count the number of files with the size of largest_file_size
      token_start_idx = 0;
      int largest_file_count = 0;
      for (int j = 0; j < cwd_fcb.size; j++)
      {
        current_byte = *get_content(fs, cwd_fcb.start_block_idx, j);
        if (current_byte == '\0')
        {
          // get this full token
          char token[21];
          my_memcpy(token, (char*)get_content(fs, cwd_fcb.start_block_idx, token_start_idx), 21);
          // get the fcb
          int fcb_idx = get_fcb_by_name(fs, token);
          FCB *fcb = fs->start_of_fcb + fcb_idx;
          if (check_fcb_on(fcb) && (fcb->size == largest_file_size))
          {
            largest_file_count++;
          }
          token_start_idx = j+1;
        }
        
      }
      printf("largest file size: %d, count: %d\n", largest_file_size, largest_file_count);

      // now we have the size, find the file or subdir with the same size and print by creation time order
      u16 last_item_time = 0;
      for (int i = 0; i < largest_file_count; i++)
      {
        // find the file with the file size of largest_file_size and the earliest created time among all unprinted items

        u16 earliest_created_time = (1<<15);
        FCB earliest_fcb;
        int earliest_fcb_idx;

        for (int j = 0; j < cwd_fcb.size; j++)
        {
          current_byte = *get_content(fs, cwd_fcb.start_block_idx, j);
          if (current_byte == '\0')
          {
            // get this full token
            char token[21];
            my_memcpy(token, (char*)get_content(fs, cwd_fcb.start_block_idx, token_start_idx), 21);
            // get the fcb
            int fcb_idx = get_fcb_by_name(fs, token);
            FCB *fcb = fs->start_of_fcb + fcb_idx;
            if (check_fcb_on(fcb) && (fcb->size == largest_file_size) && (fcb->creation_time < earliest_created_time) && (fcb->creation_time > last_item_time))
            {
              earliest_fcb = *fcb;
              earliest_created_time = fcb->creation_time;
              earliest_fcb_idx = fcb_idx;
            }
            
            token_start_idx = j+1;
          }
        }
        last_item_time = earliest_fcb.creation_time;
        if (earliest_fcb.is_dir)
        {
          printf("%s %d d\n", earliest_fcb.filename, earliest_fcb.size);
        } else {
          printf("%s %d\n", earliest_fcb.filename, earliest_fcb.size);
        }
      }
      print_count += largest_file_count;
    }
    break;
  }
  default:
    break;  // no such option
  } // end of switch
  
}


// rm, cd, mkdir, rm_rf goes here
__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
  // find the specific file in the FCB
  bool file_exists = file_exists_in_curr_dir(fs, s);
  int fcb_idx = 0;
  if (file_exists)
  {
    fcb_idx = get_fcb_by_name(fs, s);
  }
  FCB *target_fcb = fs->start_of_fcb + fcb_idx;
  

	/* Implement rm operation here */
  if (op == RM)
  {
    // delete the specific file
    if (!file_exists)
    {
      assert(0);  // file not found
    } else {
      target_check_fcb_on(fcb) = false;

      // free the content memory
      uchar *start = fs->start_of_contents + target_fcb->start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
      
      printf("fs_delete removing %d bytes of %s, start from block %d span %d\n", target_fcb->size, target_fcb->filename, target_fcb->start_block_idx, block_of_bytes(fs, target_fcb->size));

      // free the blocks  
      for (u32 i = 0; i < block_of_bytes(fs, target_fcb->size); i++)
      {
        mark_block_unused(fs, target_fcb->start_block_idx+i);
      }
      // empty the bytes, replace by 0
      for (u32 i = 0; i < target_fcb->size; i++)
      {
        start[i] = 0;
      }
    }
    
  } else if (op == MKDIR)
  {
    // create a new directory
    if (file_exists) {
      assert(0);  // directory already exists
    } else {
      // find an empty FCB
      for (int i = 0; i < fs->FCB_ENTRIES; i++)
      {
        target_fcb = &fs->start_of_fcb[i];
        if (!target_check_fcb_on(fcb))
        {
          // create a new empty directory in this file
          target_check_fcb_on(fcb) = true;
          target_fcb->size = 0;
          target_fcb->start_block_idx = 0;
          target_fcb->creation_time = gtime;
          target_fcb->modified_time = gtime;

          target_fcb->is_dir = true;
          target_fcb->parent_dir_idx = fs->cwd;  // set the parent directory to the current working directory
          target_fcb->dir_idx = i;

          // copy the dirname as filename
          int idx = 0;
          while (s[idx] != '\0')
          {
            fs->start_of_fcb[i].filename[idx] = s[idx];
            idx++;
          }
          fs->start_of_fcb[i].filename[idx] = '\0';

          set_gtime_recursive(fs, i, gtime);

          printf("create new directory fcb %s, index %d\n", s, i);
          break;
        }
        assert(0);  // no empty FCB
      }
    }
  } else if (op == CD) {
    assert(file_exists);  // if assertion failed, the directory does not exist
    fs->cwd = fcb_idx;
  } else if (op == RM_RF) {
    // Remove the app directory and all its subdirectories and files recursively
    assert(file_exists);  // if assertion failed, the directory or file does not exist

    // locate the additional fcb
    if (!target_fcb->is_dir)
    {
      // remove that regular file only
      fs_gsys(fs, RM, s);
    } else {
      // recursively remove the directory, its subdirectories and files
      // read in the contained file or directory info
      // by tokenizing

      uchar current_byte;
      int token_start_idx = 0;

      for (int i = 0; i < target_fcb->size; i++)
      {
        current_byte = *get_content(fs, target_fcb->start_block_idx, i);
        if (current_byte == '\0')
        {
          // get this full token
          char token[21];
          my_memcpy(token, (char*)get_content(fs, target_fcb->start_block_idx, token_start_idx), i-token_start_idx+1);
          fs_gsys(fs, RM_RF, token);  // recursive call to remove subdir or subfile
          token_start_idx = i+1;
        }
        
      }
      // after all subdir and subfile are removed, remove myself
      target_check_fcb_on(fcb) = false;

      // free the content memory
      uchar *start = fs->start_of_contents + target_fcb->start_block_idx * fs->STORAGE_BLOCK_SIZE; // the initial byte of the file content
      
      printf("fs_rmrf removing %d bytes of directory %s, start from block %d span %d\n", target_fcb->size, target_fcb->filename, target_fcb->start_block_idx, block_of_bytes(fs, target_fcb->size));

      // free the blocks  
      for (u32 i = 0; i < block_of_bytes(fs, target_fcb->size); i++)
      {
        mark_block_unused(fs, target_fcb->start_block_idx+i);
      }
      // empty the bytes, replace by 0
      for (u32 i = 0; i < target_fcb->size; i++)
      {
        start[i] = 0;
      }
    }
    
  }
  
  
}
